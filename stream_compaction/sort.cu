#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "sort.h"
#include <algorithm>
#include <thrust/scan.h>
#include "thrust.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
	namespace Sort {

		// Done: __global__

		__global__ void kernComputeTArray(int n, int total_falses, int* f, int *t) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;
			t[index] = index - f[index] + total_falses;
		}

		__global__ void kernComputeEArray(int n, int shift, int *e, int *in) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;
			e[index] = (in[index] >> shift) & 1 ? 0 : 1;
		}

		__global__ void kernScatter(int n, int* e, int *t, int *f, int* dev_out, int* dev_in) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;

			dev_out[!e[index] ? t[index] : f[index]] = dev_in[index];
		}

		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void sort(int n, int *odata, const int *idata, float& time) {
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int *dev_in;
			int *dev_out;
			int *e;
			int *f;
			int *t;

			//int *t_host = new int[n];

			int *e_host = new int[n];

			hipMalloc((void**)&dev_in, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_in failed!");
			hipMalloc((void**)&dev_out, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_out failed!");
			hipMalloc((void**)&e, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc e failed!");
			hipMalloc((void**)&f, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc f failed!");
			hipMalloc((void**)&t, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc t failed!");

			hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			
			float milliseconds = 0, totalTime = 0.f;
			
			//max number allowed is ilog2ceil(n) - 1; for ex: if n == 8, max value any element can have is 7
			for (int lsb = 0; lsb < 3; ++lsb) {
				hipEventRecord(start);
				//compute e array
				kernComputeEArray << <fullBlocksPerGrid, blockSize >> >(n, lsb, e, dev_in);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);
				totalTime += milliseconds;
				//scan e
				hipMemcpy(e_host, e, sizeof(int) * (n), hipMemcpyDeviceToHost);
				int total_falses = e_host[n - 1];
				thrust::exclusive_scan(e_host, e_host + n, e_host);
				total_falses += e_host[n - 1];
				hipMemcpy(f, e_host, sizeof(int) * n, hipMemcpyHostToDevice);
				hipEventRecord(start);
				//compute t array
				kernComputeTArray << <fullBlocksPerGrid, blockSize >> >(n, total_falses, f, t);

				//scatter
				kernScatter << <fullBlocksPerGrid, blockSize >> >(n, e, t, f, dev_out, dev_in);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&milliseconds, start, stop);
				totalTime += milliseconds;
				std::swap(dev_in, dev_out);
			}
			std::swap(dev_in, dev_out);
			hipMemcpy(odata, dev_out, sizeof(int) * (n), hipMemcpyDeviceToHost);
			
			time = totalTime;

			hipFree(dev_in);
			hipFree(dev_out);
			hipFree(e);
			hipFree(t);
			hipFree(f);
			
			delete[] e_host;
		}

	}
}
