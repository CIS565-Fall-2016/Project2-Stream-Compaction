#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "sort.h"
#include <algorithm>
#include <thrust/scan.h>
#include "thrust.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
	namespace Sort {

		// Done: __global__

		__global__ void kernComputeTArray(int n, int total_falses, int* f, int *t) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;
			t[index] = index - f[index] + total_falses;
		}

		__global__ void kernComputeEArray(int n, int shift, int *e, int *in) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;
			e[index] = (in[index] >> shift) & 1 ? 0 : 1;
		}

		__global__ void kernScatter(int n, int* e, int *t, int *f, int* dev_out, int* dev_in) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;

			dev_out[!e[index] ? t[index] : f[index]] = dev_in[index];
		}

		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void sort(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int *dev_in;
			int *dev_out;
			int *e;
			int *f;
			int *t;

			//int *t_host = new int[n];

			int *e_host = new int[n];

			hipMalloc((void**)&dev_in, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_in failed!");
			hipMalloc((void**)&dev_out, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_out failed!");
			hipMalloc((void**)&e, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc e failed!");
			hipMalloc((void**)&f, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc f failed!");
			hipMalloc((void**)&t, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc t failed!");

			hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			//max number allowed is ilog2ceil(n) - 1; for ex: if n == 8, max value any element can have is 7
			for (int lsb = 0; lsb < 3; ++lsb) {
				//compute e array
				kernComputeEArray << <fullBlocksPerGrid, blockSize >> >(n, lsb, e, dev_in);
				//scan e
				hipMemcpy(e_host, e, sizeof(int) * (n), hipMemcpyDeviceToHost);
				int total_falses = e_host[n - 1];
				thrust::exclusive_scan(e_host, e_host + n, e_host);
				total_falses += e_host[n - 1];
				hipMemcpy(f, e_host, sizeof(int) * n, hipMemcpyHostToDevice);
				//compute t array
				kernComputeTArray << <fullBlocksPerGrid, blockSize >> >(n, total_falses, f, t);
				//hipMemcpy(t_host, t, sizeof(int) * (n), hipMemcpyDeviceToHost);

				//scatter
				kernScatter << <fullBlocksPerGrid, blockSize >> >(n, e, t, f, dev_out, dev_in);
				std::swap(dev_in, dev_out);
			}
			std::swap(dev_in, dev_out);
			hipMemcpy(odata, dev_out, sizeof(int) * (n), hipMemcpyDeviceToHost);
			for (int i = 0; i < n; ++i)
				printf("%d\n", odata[i]);

			hipFree(dev_in);
			hipFree(dev_out);
			hipFree(e);
			hipFree(t);
			hipFree(f);
			
			delete[] e_host;
		}

	}
}
