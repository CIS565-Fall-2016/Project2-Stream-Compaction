#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#include "radix_sort.h"

namespace StreamCompaction {
	namespace RadixSort {

		__global__ void kernInitEBitMap(int N, int bit, int* inBuffer, int* outEBitMap)
		{
			const int iSelf = threadIdx.x + (blockIdx.x * blockDim.x);
			if (0 <= iSelf && iSelf < N)
			{
				outEBitMap[iSelf] = 1 - ((inBuffer[iSelf] & (1 << bit)) != 0);
			}
		}

		__global__ void kernInitTArray(int N, int numTotalFalses, int* cudaFBuffer, int* outTBuffer)
		{
			const int iSelf = threadIdx.x + (blockIdx.x * blockDim.x);
			if (0 <= iSelf && iSelf < N)
			{
				outTBuffer[iSelf] = iSelf - cudaFBuffer[iSelf] + numTotalFalses;
			}
		}

		__global__ void kernInitDArray(int N, int* cudaEBuffer, int* cudaTBuffer, int* cudaFBuffer, int* outDBuffer)
		{
			const int iSelf = threadIdx.x + (blockIdx.x * blockDim.x);
			if (0 <= iSelf && iSelf < N)
			{
				outDBuffer[iSelf] = cudaEBuffer[iSelf] ? cudaFBuffer[iSelf] : cudaTBuffer[iSelf];
			}
		}

		/**
		* Performs Parallel Radix Sort.
		*/
		void sort(int n, int *odata, const int *idata)
		{
			if (n <= 0 || odata == NULL || idata == NULL)
				return;

			const int blockSize = 128;
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int* cudaInBuffer = NULL;
			hipMalloc((void**)&cudaInBuffer, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc cudaInBuffer failed!");
			hipMemcpy(cudaInBuffer, idata, n * sizeof(int), hipMemcpyHostToDevice);

			int* cudaEBuffer = NULL;
			hipMalloc((void**)&cudaEBuffer, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc cudaEBuffer failed!");
			hipMemset(cudaEBuffer, 0, n * sizeof(int));

			int* cudaFBuffer = NULL;

			int* cudaTBuffer = NULL;
			hipMalloc((void**)&cudaTBuffer, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc cudaTBuffer failed!");
			hipMemset(cudaTBuffer, 0, n * sizeof(int));

			int* cudaDBuffer = NULL;
			hipMalloc((void**)&cudaDBuffer, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc cudaDBuffer failed!");
			hipMemset(cudaDBuffer, 0, n * sizeof(int));

			int* cudaOutBuffer = NULL;
			hipMalloc((void**)&cudaOutBuffer, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc cudaOutBuffer failed!");
			hipMemset(cudaOutBuffer, 0, n * sizeof(int));

			const int maxNumBits = ilog2ceil(n) + 1;
			for (int bitIdx = 0; bitIdx < maxNumBits; ++bitIdx)
			{
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
				kernInitEBitMap << <fullBlocksPerGrid, blockSize >> > (n, bitIdx, cudaInBuffer, cudaEBuffer);
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
				cudaFBuffer = StreamCompaction::Efficient::scanInHostPlace(n, cudaEBuffer);
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");

				int numTotalFalses = 0;
				{
					checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
					int lastEElement = 0;
					checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
					hipMemcpy(&lastEElement, cudaEBuffer + n - 1, sizeof(int), hipMemcpyDeviceToHost);
					checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
					int lastFElement = 0;
					checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
					hipMemcpy(&lastFElement, cudaFBuffer + n - 1, sizeof(int), hipMemcpyDeviceToHost);
					numTotalFalses = lastEElement + lastFElement;
				}
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
				kernInitTArray << <fullBlocksPerGrid, blockSize >> > (n, numTotalFalses, cudaFBuffer, cudaTBuffer);
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
				kernInitDArray << <fullBlocksPerGrid, blockSize >> > (n, cudaEBuffer, cudaTBuffer, cudaFBuffer, cudaDBuffer);
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
				StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, cudaOutBuffer, cudaInBuffer, cudaDBuffer);
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
				hipMemcpy(odata, cudaOutBuffer, n * sizeof(int), hipMemcpyDeviceToHost);
				hipMemcpy(cudaInBuffer, cudaOutBuffer, n * sizeof(int), hipMemcpyDeviceToDevice);
				checkCUDAErrorWithLine("hipMalloc cudaIn failed!");

				hipFree(cudaFBuffer);
			}
			

			hipFree(cudaOutBuffer);
			hipFree(cudaDBuffer);
			hipFree(cudaTBuffer);
			hipFree(cudaEBuffer);
			hipFree(cudaInBuffer);

		}

	}
}
