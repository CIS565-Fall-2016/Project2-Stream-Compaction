#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

// perform reduction
__global__ void kernScanUp(int n, int dPow, int *data) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k % dPow != 0 || k + dPow - 1 >= n)
    return;

  data[k + dPow - 1] += data[k + dPow/2 - 1];
}


// perform reduction
__global__ void kernScanDown(int n, int dPow, int *data) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k % dPow != 0 || k + dPow - 1 >= n)
    return;

  int t = data[k + dPow/2 - 1];
  data[k + dPow/2 - 1] = data[k + dPow - 1];
  data[k + dPow - 1] += t;
}

// mark nonzeroes
__global__ void kernMark(int n, int *keep, const int *data) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n)
    return;

  keep[k] = (data[k] != 0) ? 1 : 0;
}

__global__ void kernScatter(int n, int *out, const int *keep, const int *scan, const int *data) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= n)
    return;

  if (keep[k]) {
    out[scan[k]] = data[k];
  }
}

static int getPot(int n) {
  unsigned int pot = n;
  pot--;
  pot |= pot >> 1;
  pot |= pot >> 2;
  pot |= pot >> 4;
  pot |= pot >> 8;
  pot |= pot >> 16;
  pot++;

  return pot;
}

static void devScanUtil(int n, int *devData) {
  int pot  = getPot(n);

  dim3 blkDim(256);
  dim3 blkCnt((pot + blkDim.x - 1)/blkDim.x);

  int dPow = 2;
  while (dPow/2 < n) {
    kernScanUp<<<blkCnt,blkDim>>>(pot, dPow, devData);
    dPow *= 2;
  }
  hipMemset(&devData[pot-1], 0, sizeof(int));

  while (dPow > 1) {
    kernScanDown<<<blkCnt,blkDim>>>(pot, dPow, devData);
    dPow /= 2;
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
  int pot  = getPot(n);

  int *devData;
  hipMalloc((void**)&devData, pot*sizeof(int));
  hipMemset(devData, 0, pot*sizeof(int));
  hipMemcpy(devData, idata, n*sizeof(int), hipMemcpyHostToDevice);

  devScanUtil(n, devData);

  hipMemcpy(odata, devData, n*sizeof(int), hipMemcpyDeviceToHost);
  hipFree(devData);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
  int pot = getPot(n);

  // upload data
  int *devData;
  hipMalloc((void**)&devData, n*sizeof(int));
  hipMemcpy(devData, idata, n*sizeof(int), hipMemcpyHostToDevice);

  dim3 blkDim(256);
  dim3 blkCnt((n + blkDim.x - 1)/blkDim.x);

  // mark values to keep
  int *devKeep, *devScan;
  hipMalloc((void**)&devKeep, pot*sizeof(int));
  hipMemset(devKeep, 0, pot*sizeof(int));
  kernMark<<<blkCnt,blkDim>>>(n, devKeep, devData);
  hipMalloc((void**)&devScan, pot*sizeof(int));
  hipMemcpy(devScan, devKeep, pot*sizeof(int), hipMemcpyDeviceToDevice);

  // scan boolean array
  devScanUtil(n, devScan);
  int nKeep;
  hipMemcpy(&nKeep, &devScan[pot-1], sizeof(int), hipMemcpyDeviceToHost);

  // scatter to output
  int *devOut;
  hipMalloc((void**)&devOut, n*sizeof(int));
  kernScatter<<<blkCnt,blkDim>>>(n, devOut, devKeep, devScan, devData);
  hipMemcpy(odata, devOut, nKeep*sizeof(int), hipMemcpyDeviceToHost);

  hipFree(devOut);
  hipFree(devData);
  hipFree(devKeep);
  hipFree(devScan);

  return nKeep;
}

}
}
