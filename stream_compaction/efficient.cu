#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */

int *dev_Data;
int *dev_Flag;
int *dev_ScanResult;
int *dev_OutputData;
int *dev_total;

int threadPerBlock = 1024;
int BlockNum;
//__global__ void CudaUpSweep(int d, int *data, int addTimes)
//{
//	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
//	if (thid >= addTimes)
//		return;
//	int m = 1 << (d + 1);
//	if (!(thid % m))
//		data[thid + m - 1] += data[thid + (m >> 1) - 1];
//}
//
//__global__ void CudaDownSweep(int d, int *data, int addTimes)
//{
//	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
//	if (thid >= addTimes)
//		return;
//	int m = 1 << (d + 1);
//	if (!(thid % m))
//	{
//		int temp = data[thid + (m >> 1) - 1];
//		data[thid + (m >> 1) - 1] = data[thid + m - 1];
//		data[thid + m - 1] += temp;
//	}
//}
__global__ void CudaUpSweep(int d, int *data, int addTimes)
{
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thid >= addTimes)
		return;
	data[(thid + 1) * (1 << (d + 1)) - 1] += data[(thid + 1) * (1 << (d + 1)) - 1 - (1 << d)];
}

__global__ void CudaDownSweep(int d, int *data, int addTimes)
{
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thid >= addTimes)
		return;
	int m = (thid + 1) * (1 << (d + 1));
	int temp = data[m - 1 - (1 << d)];
	data[m - 1 - (1 << d)] = data[m - 1];
	data[m - 1] += temp;
}

void scan(int n, int *odata, const int *idata) {
 //   int n = 8;
	//int idata[8] ={0,1,2,3,4,5,6,7};
	//int odata[8];
	int nCeilLog = ilog2ceil(n);
	int nLength = 1 << nCeilLog;
	
	hipMalloc((void**)&dev_Data, nLength * sizeof(int));
	checkCUDAError("hipMalloc failed!");

	hipMemcpy(dev_Data, idata, sizeof(int) * nLength, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy to device failed!");

	for (int i = 0; i < nCeilLog; i++)
	{
		int addTimes = 1 << (nCeilLog - 1 - i);
		BlockNum = addTimes / threadPerBlock + 1;
		CudaUpSweep<<<BlockNum, threadPerBlock>>>(i, dev_Data, addTimes);
	}

	hipMemset(dev_Data + nLength - 1, 0, sizeof(int));
	checkCUDAError("hipMemset failed!");
	for (int i = nCeilLog - 1; i >= 0; i--)
	{
		int addTimes = 1 << (nCeilLog - 1 - i);
		BlockNum = addTimes / threadPerBlock + 1;
		CudaDownSweep<<<BlockNum, threadPerBlock>>>(i, dev_Data, addTimes);
	}
	//for (int i = 0; i < nCeilLog; i++)
	//{
	//	BlockNum = nLength / threadPerBlock + 1;
	//	CudaUpSweep<<<BlockNum, threadPerBlock>>>(i, dev_Data, nLength);
	//}

	//hipMemset(dev_Data + nLength - 1, 0, sizeof(int));
	//checkCUDAError("hipMemset failed!");
	//for (int i = nCeilLog - 1; i >= 0; i--)
	//{
	//	BlockNum = nLength / threadPerBlock + 1;
	//	CudaDownSweep<<<BlockNum, threadPerBlock>>>(i, dev_Data, nLength);
	//}

	hipMemcpy(odata, dev_Data, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy to host failed!");	
			//	for (int j = 0; j < n; j++)
		//	printf("%d ", odata[j]);
		//printf("\n");
	hipFree(dev_Data);
}

__global__ void CudaGetFlag(int *out, int *in, int n)
{
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thid >= n)
		return;
	out[thid] = in[thid] ? 1 : 0;
}

__global__ void CudaGetResult(int *result, int *flag, int *scanResult, int *data, int n)
{
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thid >= n)
		return;
	if (flag[thid])
		result[scanResult[thid]] = data[thid];
}

__global__ void CudaGetTotal(int *total, int *flag, int n)
{
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thid >= n)
		return;
	if (flag[thid])
	{
		total[0] = total[0] + total[1];
		total[1] = 100;
		printf("%d %d %d\n", thid, flag[thid], total[0]);
	}
}

void test(int *buffer, int size)
{
	int *cao = new int[size];
	hipMemcpy(cao, buffer, sizeof(int) * size, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy to host failed!");

	for (int i = 0; i < size; i++)
		printf("%d ", cao[i]);
	printf("\n");
	delete [] cao;
}
/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    // TODO
	if (n <= 0)
		return -1;
	
	int nCeilLog = ilog2ceil(n);
	int nLength = 1 << nCeilLog;

	hipMalloc((void**)&dev_Data, nLength * sizeof(int));
	hipMalloc((void**)&dev_ScanResult, nLength * sizeof(int));
	hipMalloc((void**)&dev_Flag, nLength * sizeof(int));
	hipMalloc((void**)&dev_OutputData, n * sizeof(int));
	checkCUDAError("hipMalloc failed!");

	hipMemcpy(dev_Data, idata, nLength * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy to device failed!");

	// dev_Flag is 0 or 1, calculate dev_Flag
	BlockNum = nLength / threadPerBlock + 1;
	CudaGetFlag<<<BlockNum, threadPerBlock>>>(dev_Flag, dev_Data, nLength);






	// now scan
	hipMemcpy(dev_ScanResult, dev_Flag, nLength * sizeof(int), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy device to device failed!");

float time_elapsed=0;
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord( start,0);

	for (int i = 0; i < nCeilLog; i++)
	{
		int addTimes = 1 << (nCeilLog - 1 - i);
		BlockNum = addTimes / threadPerBlock + 1;
		CudaUpSweep<<<BlockNum, threadPerBlock>>>(i, dev_ScanResult, addTimes);
	}
	hipMemset(dev_ScanResult + nLength - 1, 0, sizeof(int));
	checkCUDAError("hipMemcpy to device failed!");
	for (int i = nCeilLog - 1; i >= 0; i--)
	{
		int addTimes = 1 << (nCeilLog - 1 - i);
		BlockNum = addTimes / threadPerBlock + 1;
		CudaDownSweep<<<BlockNum, threadPerBlock>>>(i, dev_ScanResult, addTimes);
	}
	//for (int i = 0; i < nCeilLog; i++)
	//{
	//	BlockNum = nLength / threadPerBlock + 1;
	//	CudaUpSweep<<<BlockNum, threadPerBlock>>>(i, dev_ScanResult, nLength);
	//}
	//hipMemset(dev_ScanResult + nLength - 1, 0, sizeof(int));
	//checkCUDAError("hipMemcpy to device failed!");
	//for (int i = nCeilLog - 1; i >= 0; i--)
	//{
	//	BlockNum = nLength / threadPerBlock + 1;
	//	CudaDownSweep<<<BlockNum, threadPerBlock>>>(i, dev_ScanResult, nLength);
	//}
	hipEventRecord( stop,0);
hipEventSynchronize(start);
hipEventSynchronize(stop);
hipEventElapsedTime(&time_elapsed,start,stop);
//FILE* fp = fopen("efficient.txt", "a+");
//fprintf(fp, "%d %f\n", nCeilLog, time_elapsed);
//fclose(fp);
	
	BlockNum = n / threadPerBlock + 1;
	CudaGetResult<<<BlockNum, threadPerBlock>>>(dev_OutputData, dev_Flag, dev_ScanResult, dev_Data, n);
	hipMemcpy(odata, dev_OutputData, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy to host failed!");	
	
	int total, flag;
	hipMemcpy(&total, dev_ScanResult + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&flag, dev_Flag + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy device to device failed!");
	
	hipFree(dev_Data);
	hipFree(dev_ScanResult);
	hipFree(dev_Flag);
	hipFree(dev_OutputData);

	return flag ? total + 1 : total;
}

}
}
