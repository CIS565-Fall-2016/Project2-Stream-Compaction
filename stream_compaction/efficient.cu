#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
namespace Efficient {

__global__ void kernScanUpsweep(int n, int d, int * data) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= (n >> d)) {
		return;
	}
	int k = index << d;
	data[k + (1 << d) - 1] += data[k + (1 << (d - 1)) - 1];
}

__global__ void kernScanDownsweep(int n, int d, int * data) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= (n >> d)) {
		return;
	}
	int k = index << d;
	int t = data[k + (1 << d) - 1];
	data[k + (1 << d) - 1] += data[k + (1 << (d - 1)) - 1];
	data[k + (1 << (d - 1)) - 1] = t;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	int * dev_data;
	int logCeil = ilog2ceil(n);
	int nCeil = 1 << logCeil;

	hipMalloc((void**)&dev_data, nCeil * sizeof(int));
	hipMemset((void*)dev_data, 0, nCeil * sizeof(int));
	hipMemcpy((void*)dev_data, (void*)idata, n * sizeof(int), hipMemcpyHostToDevice);

#if TIMING == 1
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif

	for (int i = 1; i <= logCeil; i++) {
		int gridSize = ((nCeil >> i) + blockSize - 1) / blockSize;
		kernScanUpsweep << <gridSize, blockSize >> >(nCeil, i, dev_data);
	}

	hipMemset((void*)&dev_data[nCeil - 1], 0, sizeof(int));

	for (int i = logCeil; i >= 1; i--) {
		int gridSize = ((nCeil >> i) + blockSize - 1) / blockSize;
		kernScanDownsweep << <gridSize, blockSize >> >(nCeil, i, dev_data);
	}

#if TIMING == 1
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Efficient scan: %f milliseconds\n", milliseconds);
#endif

	hipMemcpy((void*)odata, (void*)dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_data);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	int * dev_bools;
	int * dev_idata;
	int * dev_odata;
	int * dev_indices;
	hipMalloc((void**)&dev_bools, n * sizeof(int));
	hipMalloc((void**)&dev_indices, n * sizeof(int));
	hipMalloc((void**)&dev_idata, n * sizeof(int));
	hipMalloc((void**)&dev_odata, n * sizeof(int));

	// Map to booleans
	hipMemcpy((void*)dev_idata, (void*)idata, n * sizeof(int), hipMemcpyHostToDevice);
	StreamCompaction::Common::kernMapToBoolean << <n + blockSize - 1, blockSize >> >(n, dev_bools, dev_idata);
	int * temp = (int *)malloc(n * sizeof(int));
	hipMemcpy((void*)temp, (void*)dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);

	// Do exclusive scan
	scan(n, temp, temp);
	int compactedCount = temp[n - 1] + ((idata[n - 1] == 0) ? 0 : 1);

	// Scatter
	hipMemcpy((void*)dev_indices, (void*)temp, n * sizeof(int), hipMemcpyHostToDevice);
	StreamCompaction::Common::kernScatter << <n + blockSize - 1, blockSize >> >(n, dev_odata, dev_idata, dev_bools, dev_indices);
	hipMemcpy((void*)odata, (void*)dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

	free(temp);
	hipFree(dev_bools);
	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_indices);

    return compactedCount;
}

}
}
