#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128
namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

__global__ void kernUpSweep(int n, int offset, int *buf) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int idx = (index + 1) * (offset * 2) - 1;
	if (idx >= n) return;
	//if ((index + 1) % (offset * 2) == 0) return;
	
	buf[idx] += buf[idx - offset];
	//buf[index] += buf[index - offset];
}

__global__ void kernDownSweep(int n, int offset, int *buf) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int idx = (index + 1) * (offset * 2) - 1;
	if (idx >= n) return;

	int t = buf[idx - offset];
	buf[idx - offset] = buf[idx];
	buf[idx] += t;
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	int *buf;
	int padded = 1 << ilog2ceil(n);

	hipMalloc((void**)&buf, padded * sizeof(int));
	checkCUDAError("hipMalloc buf failed!");

	hipMemcpy(buf, idata, padded * sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	int offset;
	for (int i = 0; i <= ilog2(padded); i++) {
		kernUpSweep << <fullBlocksPerGrid, blockSize >> >(padded, 1 << i, buf);
	}

	hipMemset(buf + padded - 1, 0, sizeof(int));
	for (int i = ilog2(padded); i >= 0; i--) {
		kernDownSweep << <fullBlocksPerGrid, blockSize >> >(padded, 1 << i, buf);
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, end);
	printf("Work-Efficient scan: %f ms\n", milliseconds);

	hipMemcpy(odata, buf, padded * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(buf);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	int *bools, *indices, *in, *out;
	
	hipMalloc((void**)&bools, n * sizeof(int));
	hipMalloc((void**)&indices, n * sizeof(int));
	hipMalloc((void**)&in, n * sizeof(int));
	hipMalloc((void**)&out, n * sizeof(int));

	hipMemcpy(in, idata, n * sizeof(int), hipMemcpyHostToDevice);

	float total = 0;
	float milliseconds = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, bools, in);
	
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	total += milliseconds;
	
	hipMemcpy(odata, bools, n * sizeof(int), hipMemcpyDeviceToHost);
	scan(n, odata, odata);
	int lenCompacted = odata[n - 1];
	hipMemcpy(indices, odata, n * sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, out, in, bools, indices);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	total += milliseconds;
	printf("Work-Efficient Compact: %f ms\n", total);
	hipMemcpy(odata, out, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(bools);
	hipFree(indices);
	hipFree(in);
	hipFree(out);

	return lenCompacted;
}

}
}
