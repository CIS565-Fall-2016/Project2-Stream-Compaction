#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

	__global__ void kernScanUp(int n, int d, int step, int *g_odata, int*g_idata) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index < d) {
			int ai = step * (2 * index + 1) - 1;
			int bi = step * (2 * index + 2) - 1;

			g_idata[bi] += g_idata[ai];
		}
	}

	__global__ void kernScanDown(int n, int d, int step, int *g_odata, int *g_idata) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index == 0) {
			g_idata[n - 1] = 0;
		}

		// traverse down tree & build scan  
		if (index < d) {
			int ai = step * (2 * index + 1) - 1;
			int bi = step * (2 * index + 2) - 1;


			float t = g_idata[ai];
			g_idata[ai] = g_idata[bi];
			g_idata[bi] += t;
		}
	}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + 128 - 1) / 128);
	dim3 threadsPerBlock(128);

	int* dev_in;
	int* dev_out;

	hipMalloc((void**)&dev_in, n * sizeof(int));
	checkCUDAError("hipMalloc Error dev_in.");

	hipMalloc((void**)&dev_out, n * sizeof(int));
	checkCUDAError("hipMalloc Error dev_out.");

	hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	
	for (int d = 0; d < ilog2ceil(n); d++) {
		kernScanUp << <fullBlocksPerGrid, threadsPerBlock >> >(n, d, pow(2, d+1), dev_out, dev_in);
	}

	for (int d = ilog2ceil(n); d >= 0; d--) {
		kernScanDown << <fullBlocksPerGrid, threadsPerBlock >> >(n, d, pow(2, d + 1), dev_out, dev_in);
	}
		
	hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy back failed!");

	hipFree(dev_in);
	hipFree(dev_out);
}
/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
    // TODO
    return -1;
}

}
}
