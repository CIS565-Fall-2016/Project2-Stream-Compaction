#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128
namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

__global__ void kernUpSweep(int n, int offset, int *buf) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int idx = (index + 1) * (offset * 2) - 1;
	if (idx >= n) return;
	//if ((index + 1) % (offset * 2) == 0) return;
	
	buf[idx] += buf[idx - offset];
	//buf[index] += buf[index - offset];
}

__global__ void kernDownSweep(int n, int offset, int *buf) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int idx = (index + 1) * (offset * 2) - 1;
	if (idx >= n) return;

	int t = buf[idx - offset];
	buf[idx - offset] = buf[idx];
	buf[idx] += t;
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	int *buf;
	int padded = 1 << ilog2ceil(n);

	hipMalloc((void**)&buf, padded * sizeof(int));
	checkCUDAError("hipMalloc buf failed!");

	hipMemcpy(buf, idata, padded * sizeof(int), hipMemcpyHostToDevice);

	int offset;
	for (int i = 0; i <= ilog2(padded); i++) {
		kernUpSweep << <fullBlocksPerGrid, blockSize >> >(padded, 1 << i, buf);
	}

	hipMemset(buf + padded - 1, 0, sizeof(int));
	for (int i = ilog2(padded); i >= 0; i--) {
		kernDownSweep << <fullBlocksPerGrid, blockSize >> >(padded, 1 << i, buf);
	}

	hipMemcpy(odata, buf, padded * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(buf);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	int *bools, *indices, *in, *out;
	
	hipMalloc((void**)&bools, n * sizeof(int));
	hipMalloc((void**)&indices, n * sizeof(int));
	hipMalloc((void**)&in, n * sizeof(int));
	hipMalloc((void**)&out, n * sizeof(int));

	hipMemcpy(in, idata, n * sizeof(int), hipMemcpyHostToDevice);
	StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, bools, in);
	hipMemcpy(odata, bools, n * sizeof(int), hipMemcpyDeviceToHost);
	scan(n, odata, odata);
	int lenCompacted = odata[n - 1];
	hipMemcpy(indices, odata, n * sizeof(int), hipMemcpyHostToDevice);
	StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, out, in, bools, indices);
	hipMemcpy(odata, out, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(bools);
	hipFree(indices);
	hipFree(in);
	hipFree(out);

	return lenCompacted;
}

}
}
