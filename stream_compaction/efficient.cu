#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#define MEASURE_EXEC_TIME
#include "efficient.h"
#include <vector>

#define USING_SHARED_MEMORY

#ifdef USING_SHARED_MEMORY
#define MAX_SEGMENT_SIZE 1024
#define NUM_SEG(x, ss) (((x) + (ss) - 1) / (ss))
#define ROUND_SEG_SIZE(x, ss) (NUM_SEG(x, (ss)) * (ss))

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(x) ((x) >> LOG_NUM_BANKS)
#endif

namespace StreamCompaction {
	namespace Efficient {

		int nearestMultipleOfTwo(int n)
		{
			int result = 1;
			while (result < n) result <<= 1;
			return result;
		}

#ifdef USING_SHARED_MEMORY
		__global__ void kernScan(int segSize, int * __restrict__ blockSums, int * __restrict__ odata, const int * __restrict__ idata)
		{
			extern __shared__ int temp[];

			const int base = blockIdx.x * segSize;
			const int tid = threadIdx.x;
			const int i1 = 2 * tid + 1;
			const int i2 = 2 * tid + 2;
			int offset = 1;
			int ai, bi;

			// cache data
			int gidx1 = base + tid;
			int gidx2 = gidx1 + blockDim.x;
			int lidx1 = tid + CONFLICT_FREE_OFFSET(tid);
			int lidx2 = tid + (segSize >> 1) + CONFLICT_FREE_OFFSET(tid + (segSize >> 1));
			temp[lidx1] = idata[gidx1];
			temp[lidx2] = idata[gidx2];

			// up sweep
			for (int d = segSize >> 1; d > 0; d >>= 1)
			{
				__syncthreads();

				if (tid < d)
				{
					ai = offset * i1 - 1;
					bi = offset * i2 - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);

					temp[bi] += temp[ai];
				}

				offset *= 2;
			}

			if (tid == 0)
			{
				int idx = segSize - 1 + CONFLICT_FREE_OFFSET(segSize - 1);
				if (blockSums) blockSums[blockIdx.x] = temp[idx];
				temp[idx] = 0;
			}

			// down sweep
			for (int d = 1; d < segSize; d *= 2)
			{
				offset >>= 1;
				__syncthreads();

				if (tid < d)
				{
					ai = offset * i1 - 1;
					bi = offset * i2 - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);

					int t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}

			__syncthreads();

			odata[gidx1] = temp[lidx1];
			odata[gidx2] = temp[lidx2];
		}

		__global__ void kernPerSegmentAdd(int segSize, int * __restrict__ odata, const int * __restrict__ blockSums)
		{
			int bid = blockIdx.x;
			int tid = threadIdx.x;
			int writeIdx1 = bid * segSize + 2 * tid;
			int writeIdx2 = writeIdx1 + 1;

			int sum = blockSums[bid];
			odata[writeIdx1] += sum;
			odata[writeIdx2] += sum;
		}

		inline int computeSegmentSize(int n)
		{
			return n > (MAX_SEGMENT_SIZE >> 1) ? MAX_SEGMENT_SIZE : nearestMultipleOfTwo(n);
		}

#ifdef MEASURE_EXEC_TIME
		float scanHelper(int segSize, int n, int *odata_dev, const int *idata_dev)
#else
		void scanHelper(int segSize, int n, int *odata_dev, const int *idata_dev)
#endif
		{
			// determine segment size
			int threadsPerBlock = segSize >> 1;
			int numBlocks = NUM_SEG(n, segSize);

			int *iblockSums = 0, *oblockSums = 0;
			int segSizeNextLevel;
			if (numBlocks > 1)
			{
				segSizeNextLevel = computeSegmentSize(numBlocks);
				size_t kBlockSumsSize = ROUND_SEG_SIZE(numBlocks, segSizeNextLevel) * sizeof(int);
				hipMalloc(&iblockSums, kBlockSumsSize);
				hipMalloc(&oblockSums, kBlockSumsSize);
				hipMemset(iblockSums, 0, kBlockSumsSize);
			}

#ifdef MEASURE_EXEC_TIME
			float execTime = 0.f;
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
#endif

			kernScan << <numBlocks, threadsPerBlock, (segSize + CONFLICT_FREE_OFFSET(segSize - 1)) * sizeof(int) >> >(segSize, iblockSums, odata_dev, idata_dev);

#ifdef MEASURE_EXEC_TIME
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&execTime, start, stop);
#endif

			if (numBlocks > 1)
			{
#ifdef MEASURE_EXEC_TIME
				execTime += scanHelper(segSizeNextLevel, numBlocks, oblockSums, iblockSums);

				hipEventRecord(start);
				kernPerSegmentAdd << <numBlocks, threadsPerBlock >> >(segSize, odata_dev, oblockSums);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				float et = 0.f;
				hipEventElapsedTime(&et, start, stop);
				execTime += et;
#else
				scanHelper(segSizeNextLevel, numBlocks, oblockSums, iblockSums);
				kernPerSegmentAdd << <numBlocks, threadsPerBlock >> >(segSize, odata_dev, oblockSums);
#endif

				hipFree(iblockSums);
				hipFree(oblockSums);
			}

#ifdef MEASURE_EXEC_TIME
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return execTime;
#endif
		}
#else
		__global__ void kernScanUpSweepOneLevel(int offset, int numActiveThreads, int *iodata)
		{
			int tid = blockIdx.x * blockDim.x + threadIdx.x;

			if (tid >= numActiveThreads)
			{
				return;
			}

			if (numActiveThreads == 1) // last level
			{
				iodata[2 * offset - 1] = 0;
				return;
			}

			int i1 = 2 * tid + 1;
			int i2 = i1 + 1;
			int ai, bi;

			ai = offset * i1 - 1;
			bi = offset * i2 - 1;
			iodata[bi] += iodata[ai];
		}

		__global__ void kernScanDownSweepOneLevel(int offset, int numActiveThreads, int *iodata)
		{
			int tid = blockIdx.x * blockDim.x + threadIdx.x;

			if (tid >= numActiveThreads)
			{
				return;
			}

			int i1 = 2 * tid + 1;
			int i2 = i1 + 1;
			int ai, bi;

			ai = offset * i1 - 1;
			bi = offset * i2 - 1;
			int t = iodata[ai];
			iodata[ai] = iodata[bi];
			iodata[bi] += t;
		}
#endif

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
#ifdef MEASURE_EXEC_TIME
		float scan(int n, int *odata, const int *idata)
		{
			if (n <= 0 || !odata || !idata || odata == idata)
			{
				return -1;
			}
#else
		void scan(int n, int *odata, const int *idata)
		{
			if (n <= 0 || !odata || !idata || odata == idata)
			{
				return;
			}
#endif
#ifdef USING_SHARED_MEMORY
			int segSize = computeSegmentSize(n);
			const size_t kDevArraySizeInByte = ROUND_SEG_SIZE(n, segSize) * sizeof(int);
			int *odata_dev = 0;
			int *idata_dev = 0;

			hipMalloc(&odata_dev, kDevArraySizeInByte);
			hipMalloc(&idata_dev, kDevArraySizeInByte);
			hipMemset(idata_dev, 0, kDevArraySizeInByte);
			hipMemcpy(idata_dev, idata, n * sizeof(int), hipMemcpyHostToDevice);

#ifdef MEASURE_EXEC_TIME
			float execTime = scanHelper(segSize, n, odata_dev, idata_dev);
#else
			scanHelper(segSize, n, odata_dev, idata_dev);
#endif

			hipMemcpy(odata, odata_dev, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(odata_dev);
			hipFree(idata_dev);
			hipDeviceSynchronize();

#ifdef MEASURE_EXEC_TIME
			return execTime;
#endif
#else
			const int paddedSize = nearestMultipleOfTwo(n);
			const size_t kDevArraySizeInByte = paddedSize * sizeof(int);
			int *iodata_dev = 0;

			hipMalloc(&iodata_dev, kDevArraySizeInByte);
			hipMemset(iodata_dev, 0, kDevArraySizeInByte);
			hipMemcpy(iodata_dev, idata, n * sizeof(int), hipMemcpyHostToDevice);

#ifdef MEASURE_EXEC_TIME
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
#endif

			const int threadsPerBlock = 256;
			const int numLevels = ilog2ceil(n);
			int numActiveThreads = paddedSize >> 1;
			int offset = 1;

			// up sweep
			for (int i = 0; i < numLevels; ++i)
			{
				int numBlocks = (numActiveThreads + threadsPerBlock - 1) / threadsPerBlock;
				kernScanUpSweepOneLevel << <numBlocks, threadsPerBlock >> >(offset, numActiveThreads, iodata_dev);
				numActiveThreads >>= 1;
				offset *= 2;
			}

			// down sweep
			numActiveThreads = 1;
			for (int i = 0; i < numLevels; ++i)
			{
				offset >>= 1;
				int numBlocks = (numActiveThreads + threadsPerBlock - 1) / threadsPerBlock;
				kernScanDownSweepOneLevel << <numBlocks, threadsPerBlock >> >(offset, numActiveThreads, iodata_dev);
				numActiveThreads <<= 1;
			}

#ifdef MEASURE_EXEC_TIME
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float millisceconds = 0;
			hipEventElapsedTime(&millisceconds, start, stop);
			hipEventDestroy(start);
			hipEventDestroy(stop);
#endif

			hipMemcpy(odata, iodata_dev, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(iodata_dev);
			hipDeviceSynchronize();

#ifdef MEASURE_EXEC_TIME
			return millisceconds;
#endif
#endif
		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int *odata, const int *idata)
		{
			if (n <= 0 || !odata || !idata || odata == idata)
			{
				return -1;
			}

			using StreamCompaction::Common::kernMapToBoolean;
			using StreamCompaction::Common::kernScatter;

			int *idata_dev = 0;
			int *odata_dev = 0;
			int *bools_dev = 0;
			int *indices_dev = 0;

			int segSize = computeSegmentSize(n);
			const size_t kBoolsSizeInByte = ROUND_SEG_SIZE(n, segSize) * sizeof(int);
			const size_t kIndicesSizeInByte = kBoolsSizeInByte;

			hipMalloc(&idata_dev, n * sizeof(int));
			hipMalloc(&bools_dev, kBoolsSizeInByte);
			hipMalloc(&indices_dev, kIndicesSizeInByte);

			hipMemcpy(idata_dev, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemset(bools_dev, 0, kBoolsSizeInByte);

			const int threadsPerBlock = 256;
			int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
			kernMapToBoolean << <numBlocks, threadsPerBlock >> >(n, bools_dev, idata_dev);

			scanHelper(segSize, n, indices_dev, bools_dev);

			int numElemRemained;
			hipMemcpy(&numElemRemained, indices_dev + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
			numElemRemained += idata[n - 1] ? 1 : 0;
			hipMalloc(&odata_dev, numElemRemained * sizeof(int));

			kernScatter<<<numBlocks, threadsPerBlock>>>(n, odata_dev, idata_dev, bools_dev, indices_dev);

			hipMemcpy(odata, odata_dev, numElemRemained * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(idata_dev);
			hipFree(odata_dev);
			hipFree(bools_dev);
			hipFree(indices_dev);

			return numElemRemained;
		}

	}
}
