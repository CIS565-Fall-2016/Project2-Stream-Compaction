#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

const int BlockSize = 128;


// work - efficient parallel scan 
// stride = 2^(d+1)
__global__ void kernUpSweep(int n, int stride, int *x)
{
	int k = threadIdx.x + blockDim.x * blockIdx.x;
	k *= stride;
	if (k >= n)
		return;

	int halfStride = (stride >> 1);
	x[k + stride - 1] += x[k + halfStride - 1];
}

// stride = 2^(d+1)
__global__ void kernDownSweep(int n, int stride, int *x)
{
	int k = threadIdx.x + blockDim.x * blockIdx.x;
	k *= stride;
	if (k >= n)
		return;

	int halfStride = (stride >> 1);
	int t = x[k + halfStride - 1];
	x[k + halfStride - 1] = x[k + stride - 1];
	x[k + stride - 1] += t;
}

// helper function for set data[index] = 0
__global__ void kernSetZero(int index, int* data)
{
	data[index] = 0;
}

// exclusive scan data is pointer on device
void scan_device(int n, int *data)
{
	int maxD = ilog2ceil(n) - 1;

	// up
	int stride;

	for (int d = 0; d <= maxD; ++d)
	{
		stride = (1 << (d + 1));
		int blockNumber = (n / stride + BlockSize - 1) / BlockSize;
		kernUpSweep << <blockNumber, BlockSize >> >(n, stride, data);
	}

	// set last to zero !
	kernSetZero << <1, 1 >> >(n - 1, data);

	// down
	for (int d = maxD; d >= 0; d--)
	{
		stride = (1 << (d + 1));
		int blockNumber = (n / stride + BlockSize - 1) / BlockSize;
		kernDownSweep << <blockNumber, BlockSize >> >(n, stride, data);
	}

	checkCUDAError("fk");
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) 
{

	int bufferSize = (1 << ilog2ceil(n));
	int *dev_buffer;

	hipMalloc((void**)&dev_buffer, sizeof(int)*bufferSize);
	checkCUDAError("hipMalloc dev failed");

	hipMemset(dev_buffer, 0, bufferSize*sizeof(int));
	hipMemcpy(dev_buffer, idata, sizeof(int)*n, hipMemcpyHostToDevice);

	scan_device(bufferSize, dev_buffer);

	hipMemcpy(odata, dev_buffer, sizeof(int)*n, hipMemcpyDeviceToHost);

	hipFree(dev_buffer);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {

	int bufferSize = (1 << ilog2ceil(n));

	int *dev_input, *dev_bools, *dev_indices, *dev_output;

	hipMalloc((void**)&dev_input, n * sizeof(int));
	checkCUDAError("hipMalloc dev_input failed");

	hipMalloc((void**)&dev_bools, bufferSize * sizeof(int));
	checkCUDAError("hipMalloc dev_bools failed");

	hipMalloc((void**)&dev_indices, bufferSize * sizeof(int));
	checkCUDAError("hipMalloc dev_indices failed");

	hipMalloc((void**)&dev_output, n * sizeof(int));
	checkCUDAError("hipMalloc dev_out failed");

	hipMemcpy(dev_input, idata, n*sizeof(int), hipMemcpyHostToDevice);
	
	// map to booleans 
	hipMemset(dev_bools, 0, bufferSize*sizeof(int));
	dim3 blocks((n + BlockSize - 1) / BlockSize);
	Common::kernMapToBoolean << <blocks, BlockSize >> >(n, dev_bools, dev_input);

	hipMemcpy(dev_indices, dev_bools, bufferSize*sizeof(int), hipMemcpyDeviceToDevice);
	
	// run scan
	scan_device(bufferSize, dev_indices);

	// scatter 
	Common::kernScatter << <blocks, BlockSize >> >(n, dev_output, dev_input, dev_bools, dev_indices);

	// get length
	int len;
	hipMemcpy(&len, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);

	if (idata[n - 1] != 0) // last element
	{
		len++;
	}

	// copy result to odata
	hipMemcpy(odata, dev_output, len*sizeof(int), hipMemcpyDeviceToHost);
	

	// free memory
	hipFree(dev_input);
	hipFree(dev_bools);
	hipFree(dev_indices);
	hipFree(dev_output);

    return len;
}

}
}
