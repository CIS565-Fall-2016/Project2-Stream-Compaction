#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#define MEASURE_EXEC_TIME
#include "efficient.h"
#include <vector>


namespace StreamCompaction {
	namespace Efficient {

#ifdef USING_SHARED_MEMORY
		__global__ void kernScan(int segSize, int * __restrict__ blockSums, int * __restrict__ odata, const int * __restrict__ idata)
		{
			extern __shared__ int temp[];

			const int base = blockIdx.x * segSize;
			const int tid = threadIdx.x;
			const int i1 = 2 * tid + 1;
			const int i2 = 2 * tid + 2;
			int offset = 1;
			int ai, bi;

			// cache data
			int gidx1 = base + tid;
			int gidx2 = gidx1 + blockDim.x;
			int lidx1 = tid + CONFLICT_FREE_OFFSET(tid);
			int lidx2 = tid + (segSize >> 1) + CONFLICT_FREE_OFFSET(tid + (segSize >> 1));
			temp[lidx1] = idata[gidx1];
			temp[lidx2] = idata[gidx2];

			// up sweep
			for (int d = segSize >> 1; d > 0; d >>= 1)
			{
				__syncthreads();

				if (tid < d)
				{
					ai = offset * i1 - 1;
					bi = offset * i2 - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);

					temp[bi] += temp[ai];
				}

				offset *= 2;
			}

			if (tid == 0)
			{
				int idx = segSize - 1 + CONFLICT_FREE_OFFSET(segSize - 1);
				if (blockSums) blockSums[blockIdx.x] = temp[idx];
				temp[idx] = 0;
			}

			// down sweep
			for (int d = 1; d < segSize; d *= 2)
			{
				offset >>= 1;
				__syncthreads();

				if (tid < d)
				{
					ai = offset * i1 - 1;
					bi = offset * i2 - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);

					int t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}

			__syncthreads();

			odata[gidx1] = temp[lidx1];
			odata[gidx2] = temp[lidx2];
		}

		__global__ void kernPerSegmentAdd(int segSize, int * __restrict__ odata, const int * __restrict__ blockSums)
		{
			int bid = blockIdx.x;
			int tid = threadIdx.x;
			int writeIdx1 = bid * segSize + 2 * tid;
			int writeIdx2 = writeIdx1 + 1;

			int sum = blockSums[bid];
			odata[writeIdx1] += sum;
			odata[writeIdx2] += sum;
		}

		void scanHelper(int segSize, int n, int *odata_dev, const int *idata_dev)
		{
			// determine segment size
			int threadsPerBlock = segSize >> 1;
			int numBlocks = NUM_SEG(n, segSize); // also numSegs

			int *iblockSums = 0, *oblockSums = 0;
			int segSizeNextLevel;
			if (numBlocks > 1)
			{
				segSizeNextLevel = computeSegmentSize(numBlocks);
				size_t offsetInDW = alignedSize(numBlocks * segSize * sizeof(int), 256) >> 2;
				iblockSums = const_cast<int *>(idata_dev + offsetInDW);
				oblockSums = odata_dev + offsetInDW;
			}

			kernScan << <numBlocks, threadsPerBlock, (segSize + CONFLICT_FREE_OFFSET(segSize - 1)) * sizeof(int) >> >(segSize, iblockSums, odata_dev, idata_dev);

			if (numBlocks > 1)
			{
				scanHelper(segSizeNextLevel, numBlocks, oblockSums, iblockSums);
				kernPerSegmentAdd << <numBlocks, threadsPerBlock >> >(segSize, odata_dev, oblockSums);
			}
		}
#else
		__global__ void kernScanUpSweepOneLevel(int offset, int numActiveThreads, int *iodata)
		{
			int tid = blockIdx.x * blockDim.x + threadIdx.x;

			if (tid >= numActiveThreads)
			{
				return;
			}

			if (numActiveThreads == 1) // last level
			{
				iodata[2 * offset - 1] = 0;
				return;
			}

			int i1 = 2 * tid + 1;
			int i2 = i1 + 1;
			int ai, bi;

			ai = offset * i1 - 1;
			bi = offset * i2 - 1;
			iodata[bi] += iodata[ai];
		}

		__global__ void kernScanDownSweepOneLevel(int offset, int numActiveThreads, int *iodata)
		{
			int tid = blockIdx.x * blockDim.x + threadIdx.x;

			if (tid >= numActiveThreads)
			{
				return;
			}

			int i1 = 2 * tid + 1;
			int i2 = i1 + 1;
			int ai, bi;

			ai = offset * i1 - 1;
			bi = offset * i2 - 1;
			int t = iodata[ai];
			iodata[ai] = iodata[bi];
			iodata[bi] += t;
		}
#endif

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
#ifdef MEASURE_EXEC_TIME
		float scan(int n, int *odata, const int *idata)
		{
			if (n <= 0 || !odata || !idata || odata == idata)
			{
				return -1;
			}
#else
		void scan(int n, int *odata, const int *idata)
		{
			if (n <= 0 || !odata || !idata || odata == idata)
			{
				return;
			}
#endif
#ifdef USING_SHARED_MEMORY
			int segSize = computeSegmentSize(n);
			const size_t kDevArraySizeInByte = computeActualMemSize<int>(n);
			int *odata_dev = 0;
			int *idata_dev = 0;

			hipMalloc(&odata_dev, kDevArraySizeInByte);
			hipMalloc(&idata_dev, kDevArraySizeInByte);
			hipMemset(idata_dev, 0, kDevArraySizeInByte);
			hipMemcpy(idata_dev, idata, n * sizeof(int), hipMemcpyHostToDevice);

#ifdef MEASURE_EXEC_TIME
			float execTime = 0.f;
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
#endif

			scanHelper(segSize, n, odata_dev, idata_dev);

#ifdef MEASURE_EXEC_TIME
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&execTime, start, stop);
			hipEventDestroy(start);
			hipEventDestroy(stop);
#endif

			hipMemcpy(odata, odata_dev, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(odata_dev);
			hipFree(idata_dev);
			hipDeviceSynchronize();

#ifdef MEASURE_EXEC_TIME
			return execTime;
#endif
#else
			const int paddedSize = nearestMultipleOfTwo(n);
			const size_t kDevArraySizeInByte = paddedSize * sizeof(int);
			int *iodata_dev = 0;

			hipMalloc(&iodata_dev, kDevArraySizeInByte);
			hipMemset(iodata_dev, 0, kDevArraySizeInByte);
			hipMemcpy(iodata_dev, idata, n * sizeof(int), hipMemcpyHostToDevice);

#ifdef MEASURE_EXEC_TIME
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
#endif

			const int threadsPerBlock = 256;
			const int numLevels = ilog2ceil(n);
			int numActiveThreads = paddedSize >> 1;
			int offset = 1;

			// up sweep
			for (int i = 0; i < numLevels; ++i)
			{
				int numBlocks = (numActiveThreads + threadsPerBlock - 1) / threadsPerBlock;
				kernScanUpSweepOneLevel << <numBlocks, threadsPerBlock >> >(offset, numActiveThreads, iodata_dev);
				numActiveThreads >>= 1;
				offset *= 2;
			}

			// down sweep
			numActiveThreads = 1;
			for (int i = 0; i < numLevels; ++i)
			{
				offset >>= 1;
				int numBlocks = (numActiveThreads + threadsPerBlock - 1) / threadsPerBlock;
				kernScanDownSweepOneLevel << <numBlocks, threadsPerBlock >> >(offset, numActiveThreads, iodata_dev);
				numActiveThreads <<= 1;
			}

#ifdef MEASURE_EXEC_TIME
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float millisceconds = 0;
			hipEventElapsedTime(&millisceconds, start, stop);
			hipEventDestroy(start);
			hipEventDestroy(stop);
#endif

			hipMemcpy(odata, iodata_dev, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(iodata_dev);
			hipDeviceSynchronize();

#ifdef MEASURE_EXEC_TIME
			return millisceconds;
#endif
#endif
		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
#ifdef MEASURE_EXEC_TIME
		int compact(int n, int *odata, const int *idata, float *pExecTime)
#else
		int compact(int n, int *odata, const int *idata)
#endif
		{
			if (n <= 0 || !odata || !idata || odata == idata)
			{
				return -1;
			}

			using StreamCompaction::Common::kernMapToBoolean;
			using StreamCompaction::Common::kernScatter;

#ifdef MEASURE_EXEC_TIME
			float &execTime = *pExecTime;

			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
#endif

			int *idata_dev = 0;
			int *odata_dev = 0;
			int *bools_dev = 0;
			int *indices_dev = 0;

			int segSize = computeSegmentSize(n);
			const size_t kBoolsSizeInByte = computeActualMemSize<int>(n);
			const size_t kIndicesSizeInByte = kBoolsSizeInByte;

			hipMalloc(&idata_dev, n * sizeof(int));
			hipMalloc(&bools_dev, kBoolsSizeInByte);
			hipMalloc(&indices_dev, kIndicesSizeInByte);

			hipMemcpy(idata_dev, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemset(bools_dev, 0, kBoolsSizeInByte);

			const int threadsPerBlock = 256;
			int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

#ifdef MEASURE_EXEC_TIME
			hipEventRecord(start);

			kernMapToBoolean << <numBlocks, threadsPerBlock >> >(n, bools_dev, idata_dev);

			scanHelper(segSize, n, indices_dev, bools_dev);

			int numElemRemained;
			hipMemcpy(&numElemRemained, indices_dev + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
			numElemRemained += idata[n - 1] ? 1 : 0;
			hipMalloc(&odata_dev, numElemRemained * sizeof(int));

			kernScatter<<<numBlocks, threadsPerBlock>>>(n, odata_dev, idata_dev, bools_dev, indices_dev);

			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&execTime, start, stop);
#else
			kernMapToBoolean << <numBlocks, threadsPerBlock >> >(n, bools_dev, idata_dev);

			scanHelper(segSize, n, indices_dev, bools_dev);

			int numElemRemained;
			hipMemcpy(&numElemRemained, indices_dev + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
			numElemRemained += idata[n - 1] ? 1 : 0;
			hipMalloc(&odata_dev, numElemRemained * sizeof(int));

			kernScatter << <numBlocks, threadsPerBlock >> >(n, odata_dev, idata_dev, bools_dev, indices_dev);
#endif

			hipMemcpy(odata, odata_dev, numElemRemained * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(idata_dev);
			hipFree(odata_dev);
			hipFree(bools_dev);
			hipFree(indices_dev);

			return numElemRemained;
		}

	}
}
