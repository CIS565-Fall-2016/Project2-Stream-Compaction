#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "stdio.h"
#include "stdlib.h"

#define blockSize 256
#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
	namespace Efficient {

		__device__ int threadIndex() {
			return (blockIdx.x * blockDim.x) + threadIdx.x;
		}

		__global__ void kernUpSweep(int n, int d, int *odata, int *idata) {
			int index = threadIndex();
			if (index >= n) return;
			int addTerm = (index + 1) % (d * 2) == 0 ? idata[index - d] : 0;
			odata[index] = idata[index] + addTerm;
		}

		__global__ void kernDownSweep(int length, int d, int *odata, int *idata) {
			int index = threadIndex();
			if (index >= length) return;

			// On the first iteration, and using only one thread, set the last element to 0.
			if ((index + 1) % d == 0) {
				int swapIndex = index - (d / 2);
				int term = (length == d) && (index == d - 1) ? 0 : idata[index];
				odata[index] = term + idata[swapIndex];
				odata[swapIndex] = term;
			}
		}

		int bufferToPow2(int n) {
			return pow(2, ceil(log2(n))); // n rounded up to the nearest power of 2 
		}

		void dev_scan(int n, int *dev_odata, int *dev_idata) {

			int bufferedLength = bufferToPow2(n);
			int numBlocks = getNumBlocks(blockSize, n); // enough blocks to allocate one thread to each array element

			// upsweep
			for (int d = 1; d <= n; d *= 2) {
				kernUpSweep << <numBlocks, blockSize >> >(n, d, dev_odata, dev_idata);

				// swap dev_idata with dev_odata
				int *swap = dev_idata;
				dev_idata = dev_odata;
				dev_odata = swap;
			}

			// downsweep
			for (int d = bufferedLength; d >= 1; d /= 2) {
				kernDownSweep << <numBlocks, blockSize >> >(bufferedLength, d, dev_odata, dev_idata);

				// swap dev_idata with dev_odata
				int *swap = dev_idata;
				dev_idata = dev_odata;
				dev_odata = swap;
			}
		}


		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int *odata, const int *idata) {

			// declare arrays
			int* dev_idata;
			int* dev_odata;

			int bufferedLength = bufferToPow2(n);

			// allocate memory
			hipMalloc((void**)&dev_idata, bufferedLength * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, bufferedLength * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");

			// copy memory and run the algorithm
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			dev_scan(n, dev_odata, dev_idata);

			hipMemcpy(odata, dev_idata, n* sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_idata);
			hipFree(dev_odata);
		}

		/**
		* Performs stream compaction on idata, storing the result into odata.
		* All zeroes are discarded.
		*
		* @param n      The number of elements in idata.
		* @param odata  The array into which to store elements.
		* @param idata  The array of elements to compact.
		* @returns      The number of elements remaining after compaction.
		*/
		int compact(int n, int *odata, const int *idata) {
			// declare arrays
			int* dev_idata;
			int* dev_odata;
			int* dev_bools;
			int* dev_pingPong;
			int* dev_indices;
			int* bools = (int*)calloc(n, sizeof(int));
			int* indices = (int*)calloc(n, sizeof(int));
			int* pingPong = (int*)calloc(n, sizeof(int));

			//hipEvent_t start, stop;
			//hipEventCreate(&start);
			//hipEventCreate(&stop);

			//hipEventRecord(start);
			//saxpy << <(N + 255) / 256, 256 >> >(N, 2.0f, d_x, d_y);
			//hipEventRecord(stop);

			//hipEventSynchronize(stop);6
			//float milliseconds = 0;
			//hipEventElapsedTime(&milliseconds, start, stop);

			// allocate memory
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_bools, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_pingPong, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_pingPong failed!");
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_indices failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_odata failed!");

			// copy input data to device
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			////////////
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
			////////////////

			// enough blocks to allocate one thread to each array element
			int numBlocks = (n / blockSize) + 1;

			// get array of booleans determining whether 
			Common::kernMapToBoolean << <numBlocks, blockSize >> > (n, dev_bools, dev_idata);
			hipMemcpy(dev_pingPong, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);

			// allocate memory and run scan
			dev_scan(n, dev_indices, dev_pingPong);

			Common::kernScatter << <numBlocks, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);

			///////////
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			printf("%f\n", milliseconds);
			hipEventDestroy(start);
			hipEventDestroy(stop);
			/////////


			// copy from device
			hipMemcpy(indices, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
			int newLength = indices[n - 1] + bools[n - 1]; // return value
			hipMemcpy(odata, dev_odata, newLength * sizeof(int), hipMemcpyDeviceToHost);

			// free memory
			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_bools);
			hipFree(dev_indices);
			free(indices);
			free(bools);
			free(pingPong);

			return newLength;
		}

	}
}
