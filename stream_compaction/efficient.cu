#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

__global__ void upSweep(int n, int d, int *data) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	int prevOffset = d == 0 ? 1 : 2 << (d - 1);
	int offset = prevOffset * 2;

	if (index < n && index % offset == 0) {
		data[index + offset - 1] += data[index + prevOffset - 1];
	}
}

__global__ void downSweep(int n, int d, int *data) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	int prevOffset = d == 0 ? 1 : 2 << (d - 1);
	int offset = prevOffset * 2;

	if (index < n && index % offset == 0) {
		int t = data[index + prevOffset - 1];
		data[index + prevOffset - 1] = data[index + offset - 1];
		data[index + offset - 1] += t;
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	int blockSize = 128;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	
	int nearestPow = 2 << (ilog2ceil(n) - 1); //assume n > 0

	int* dev_data;
	hipMalloc((void**)&dev_data, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_data failed!");
	hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

	// Up-sweep
	int numLevels = ilog2ceil(nearestPow) - 1;
	for (int d = 0; d <= numLevels; d++) {
		upSweep << <fullBlocksPerGrid, blockSize >> >(nearestPow, d, dev_data);
	}

	hipMemcpy(odata, dev_data, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
	//printf("AFTER UPSWEEP: [\n");
	//for (int i = 0; i < nearestPow; i++) {
	//	printf("%d\n", odata[i]);
	//}
	//printf("]\n");
	odata[nearestPow - 1] = 0;
	hipMemcpy(dev_data, odata, sizeof(int) * nearestPow, hipMemcpyHostToDevice);

	//Down-sweep
	for (int d = numLevels; d >= 0; d--) {
		//printf("LEVEL: %d\n", d);
		//hipMemcpy(odata, dev_data, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
		//printf("[ ");
		//for (int i = 0; i < nearestPow; i++) {
		//	printf("%d ", odata[i]);
		//}
		//printf("]\n");
		downSweep << <fullBlocksPerGrid, blockSize >> >(nearestPow, d, dev_data);
	}

	hipMemcpy(odata, dev_data, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
	//printf("AFTER DOWNSWEEP: [\n");
	//for (int i = 0; i < nearestPow; i++) {
	//	printf("%d\n", odata[i]);
	//}
	//printf("]\n");
	
	hipFree(dev_data);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	int blockSize = 128;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

	int nearestPow = 2 << (ilog2ceil(n) - 1); //assume n > 0

	int* dev_idata;
	int* dev_odata;
	int* dev_bools;
	int* dev_indices;
	int* bools;
	int* indices;
	
	hipMalloc((void**)&dev_idata, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_idata failed!");
	hipMemset(dev_idata, 0, nearestPow);
	hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_odata, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_odata failed!");

	hipMalloc((void**)&dev_bools, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_bools failed!");
	bools = (int*)malloc(nearestPow * sizeof(int));

	hipMalloc((void**)&dev_indices, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_indices failed!");
	indices = (int*)malloc(nearestPow * sizeof(int));

	StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(nearestPow, dev_bools, dev_idata);
	hipMemcpy(bools, dev_bools, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
	//printf("BOOLS: [\n");
	//for (int i = 0; i < nearestPow; i++) {
	//	printf("%d\n", bools[i]);
	//}
	//printf("]\n");

	scan(n, indices, bools);
	//printf("INDICES: [\n");
	//for (int i = 0; i < nearestPow; i++) {
	//	printf("%d\n", indices[i]);
	//}
	//printf("]\n");
	hipMemcpy(dev_indices, indices, sizeof(int) * nearestPow, hipMemcpyHostToDevice);

	StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(nearestPow, dev_odata, dev_idata, dev_bools, dev_indices);

	hipMemcpy(indices, dev_indices, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
	int j = nearestPow - 1;
	do {
		j--;
	} while (indices[j] == indices[j + 1]);
	int compactLength = indices[j] + 1;
	//printf("COMPACT LENGTH:%d\n", compactLength);
	hipMemcpy(odata, dev_odata, sizeof(int) * compactLength, hipMemcpyDeviceToHost);
	//printf("RESULT: [\n");
	//for (int i = 0; i < compactLength; i++) {
	//	printf("%d\n", odata[i]);
	//}
	//printf("]\n");

	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_bools);
	hipFree(dev_indices);
	free(bools);
	free(indices);

	return compactLength;
}

}
}
