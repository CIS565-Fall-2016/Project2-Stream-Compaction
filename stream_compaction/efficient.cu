#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

__global__ void upSweep(int n, int d, int *data, bool isRoot) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= n) return;

	if (isRoot) {
		data[n - 1] = 0;
	}
	else {
		int prevOffset = d == 0 ? 1 : 2 << (d - 1);
		int offset = prevOffset * 2;

		if (index % offset == 0) {
			data[index + offset - 1] += data[index + prevOffset - 1];
		}
	}
}

__global__ void downSweep(int n, int d, int *data) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	int prevOffset = d == 0 ? 1 : 2 << (d - 1);
	int offset = prevOffset * 2;

	if (index < n && index % offset == 0) {
		int t = data[index + prevOffset - 1];
		data[index + prevOffset - 1] = data[index + offset - 1];
		data[index + offset - 1] += t;
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
float scan(int n, int *odata, const int *idata) {
	int blockSize = 128;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	
	int nearestPow = 2 << (ilog2ceil(n) - 1); //assume n > 0

	int* dev_data;
	hipMalloc((void**)&dev_data, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_data failed!");
	hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// Up-sweep
	int numLevels = ilog2ceil(nearestPow);
	for (int d = 0; d < numLevels; d++) {
		upSweep << <fullBlocksPerGrid, blockSize >> >(nearestPow, d, dev_data, d == (numLevels - 1));
	}

	//Down-sweep
	for (int d = numLevels; d >= 0; d--) {
		downSweep << <fullBlocksPerGrid, blockSize >> >(nearestPow, d, dev_data);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(odata, dev_data, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
	
	hipFree(dev_data);
	return milliseconds;
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata, float* timer) {
	int blockSize = 128;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

	int nearestPow = 2 << (ilog2ceil(n) - 1); //assume n > 0

	int* dev_idata;
	int* dev_odata;
	int* dev_bools;
	int* dev_indices;
	int* indices;
	
	hipMalloc((void**)&dev_idata, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_idata failed!");
	hipMemset(dev_idata, 0, nearestPow);
	hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_odata, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_odata failed!");

	hipMalloc((void**)&dev_bools, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_bools failed!");

	hipMalloc((void**)&dev_indices, nearestPow * sizeof(int));
	checkCUDAError("hipMalloc dev_indices failed!");
	indices = (int*)malloc(nearestPow * sizeof(int));

	hipEvent_t start, stop;
	if (timer) {
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);
	}

	StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(nearestPow, dev_bools, dev_idata);
	hipMemcpy(dev_indices, dev_bools, sizeof(int) * nearestPow, hipMemcpyDeviceToDevice);

	// Up-sweep
	int numLevels = ilog2ceil(nearestPow);
	for (int d = 0; d < numLevels; d++) {
		upSweep << <fullBlocksPerGrid, blockSize >> >(nearestPow, d, dev_indices, d == (numLevels - 1));
	}

	//Down-sweep
	for (int d = numLevels; d >= 0; d--) {
		downSweep << <fullBlocksPerGrid, blockSize >> >(nearestPow, d, dev_indices);
	}

	StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(nearestPow, dev_odata, dev_idata, dev_bools, dev_indices);
	
	if (timer) {
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		(*timer) += milliseconds;
	}

	hipMemcpy(indices, dev_indices, sizeof(int) * nearestPow, hipMemcpyDeviceToHost);
	int j = nearestPow - 1;
	do {
		j--;
	} while (indices[j] == indices[j + 1]);
	int compactLength = indices[j] + 1;

	hipMemcpy(odata, dev_odata, sizeof(int) * compactLength, hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_bools);
	hipFree(dev_indices);
	free(indices);

	return compactLength;
}

}
}
