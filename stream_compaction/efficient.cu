#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

	__global__ void kernDownSweep(int d, int n, int* idata) {
		int index = threadIdx.x + (blockDim.x * blockIdx.x);

		if (index >= n) return;
		
		int off_n = 1 << (d + 1);
		int off = (1 << d);
		if ((index % (off_n) == 0)) {
			int temp = idata[index + (1 << d) - 1];
			idata[index + off - 1] = idata[index + off_n - 1];
			idata[index + off_n - 1] += temp;	
		}
	}

	__global__ void kernUpSweep(int d, int n, int* idata) {
		int index = threadIdx.x + (blockDim.x * blockIdx.x);

		if (index >= n) return;
		int off_n = 1 << (d + 1);
		int off = (1 << d);
		if (index % off_n == 0)
			idata[index + off_n - 1] += idata[index + off - 1];
	}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata, float& time) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//for non power of 2
	int ilog = ilog2ceil(n);
	int off_n = 1 << ilog;

	dim3 fullBlocksPerGrid((off_n + blockSize - 1) / blockSize);
	
	int *dev_in;

	hipMalloc((void**)&dev_in, off_n * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_in failed!");

	hipMemcpy(dev_in, idata, sizeof(int) * off_n, hipMemcpyHostToDevice);
	float milliseconds = 0;
	hipEventRecord(start);
	//up-sweep
	for (int d = 0; d < ilog; ++d) {
		kernUpSweep << <fullBlocksPerGrid, blockSize >> >(d, off_n, dev_in);
	}
	
	//set the last value as zero
	hipMemset(dev_in + (off_n - 1), 0, sizeof(int));
	
	//down-sweep
	for (int d = ilog - 1; d >= 0; --d) {
		kernDownSweep << <fullBlocksPerGrid, blockSize >> >(d, off_n, dev_in);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	time = milliseconds;
	hipMemcpy(odata, dev_in, sizeof(int) * (n), hipMemcpyDeviceToHost);
	
	hipFree(dev_in);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata, float& time) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

	int *bools;
	int *dev_in;
	int *dev_out;
	int *indices;
	int *tmp = new int[n];
	int *tmp_bools = new int[n];
	hipMalloc((void**)&dev_in, n * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_in failed!");
	hipMalloc((void**)&dev_out, n * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_out failed!");
	hipMalloc((void**)&indices, n * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc indices failed!");
	hipMalloc((void**)&bools, n * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc bools failed!");

	hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	float milliseconds = 0, totalTime = 0.f;
	hipEventRecord(start);
	//map to boolean
	Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, bools, dev_in);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totalTime += milliseconds;
	int j = 0;
	//scan
	hipMemcpy(odata, bools, sizeof(int) * (n), hipMemcpyDeviceToHost);
	float time_scan;
	scan(n, tmp, odata, time_scan);
	totalTime += time_scan;
	hipMemcpy(indices, tmp, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(tmp_bools, bools, sizeof(int) * (n), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; ++i) {
		j = tmp_bools[i] == 1 ? j + 1 : j;
	}
	hipEventRecord(start);
	//scatter
	Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_out, dev_in, bools, indices);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totalTime += milliseconds;

	time = totalTime;
	hipMemcpy(odata, dev_out, sizeof(int) * (n), hipMemcpyDeviceToHost);
	
	hipFree(dev_in);
	hipFree(bools);
	hipFree(dev_out);
	hipFree(indices);

	delete[] tmp;
	delete[] tmp_bools;
    return n - j;
}

}
}
