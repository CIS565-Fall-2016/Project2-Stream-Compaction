#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128
namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

__global__ void kernUpSweep(int n, int offset, int *buf) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= (n >> offset)) return;
	int idx = index << offset;
	buf[idx + (1 << offset) - 1] += buf[idx + (1 << (offset - 1)) - 1];
}

__global__ void kernDownSweep(int n, int offset, int *buf) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= (n >> offset)) return;
	int idx = index << offset;
	int t = buf[idx + (1 << offset) - 1];
	buf[idx + (1 << offset) - 1] += buf[idx + (1 << (offset - 1)) - 1];
	buf[idx + (1 << (offset - 1)) - 1] = t;
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	
	int *buf;
	int padded = 1 << ilog2ceil(n);

	hipMalloc((void**)&buf, padded * sizeof(int));
	hipMemcpy(buf, idata, n * sizeof(int), hipMemcpyHostToDevice);

	int offset;
	int fullBlocksPerGrid = 0;
	float total = 0;
	float milliseconds = 0;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	for (int i = 1; i <= ilog2(padded); i++) {
		fullBlocksPerGrid = ((padded >> i) + blockSize - 1) / blockSize;
		kernUpSweep << <fullBlocksPerGrid, blockSize >> >(padded, i, buf);
	}
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	total += milliseconds;

	hipMemset(buf + padded - 1, 0, sizeof(int));
	
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	for (int i = ilog2(padded); i >= 1; i--) {
		fullBlocksPerGrid = ((padded >> i) + blockSize - 1) / blockSize;
		kernDownSweep << <fullBlocksPerGrid, blockSize >> >(padded, i, buf);
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	total += milliseconds;
	printf("Work-Efficient scan: %f ms\n", total);

	hipMemcpy(odata, buf, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(buf);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	int *bools, *indices, *in, *out;
	
	hipMalloc((void**)&bools, n * sizeof(int));
	hipMalloc((void**)&indices, n * sizeof(int));
	hipMalloc((void**)&in, n * sizeof(int));
	hipMalloc((void**)&out, n * sizeof(int));

	hipMemcpy(in, idata, n * sizeof(int), hipMemcpyHostToDevice);

	float total = 0;
	float milliseconds = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, bools, in);
	
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	total += milliseconds;
	
	hipMemcpy(odata, bools, n * sizeof(int), hipMemcpyDeviceToHost);
	scan(n, odata, odata);
	int lenCompacted = odata[n - 1];
	hipMemcpy(indices, odata, n * sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, out, in, bools, indices);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&milliseconds, start, end);
	total += milliseconds;
	printf("Work-Efficient Compact: %f ms\n", total);
	hipMemcpy(odata, out, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(bools);
	hipFree(indices);
	hipFree(in);
	hipFree(out);

	return lenCompacted;
}

}
}
