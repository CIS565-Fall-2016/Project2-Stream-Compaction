#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include "common.h"
#include "radix.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Radix {

	//b array
	__global__ void kernTestTrueFalseOnRightKthBit(int n, int k, int* odata, const int* idata) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n) {
			return;
		}
		if (((1 << k) & idata[index]) == (1<<k)) {
			odata[index] = 1;
		} else {
			odata[index] = 0;
		}

	}

	//e array(! operation)
	__global__ void kernNotOperatorOnArray(int n, int *odata, const int *idata) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index >= n) {
			return;
		}
		odata[index] = 1 - idata[index];
	}

	//t array
	__global__ void kernComputeTArray(int n, const int *bArray, const int *idata, int *odata) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index >= n) {
			return;
		}
		odata[index] = index - idata[index] + idata[n - 1] + !bArray[n - 1];
	}

	//d array
	__global__ void kernComputeDArray(int n, int * dArray, const int *bArray, const int *fArray, const int *tArray) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index >= n) {
			return;
		}
		dArray[index] = (bArray[index] ? tArray[index] : fArray[index]);
	}

	//Reshuffle Index
	__global__ void kernReshuffle(int n, int* idata, int *odata, const int *dArray) {
		int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index >= n) {
			return;
		}
		odata[dArray[index]] = idata[index];
	}

	void RadixSort(int n, int* idata, int maxNum) {
		int *devIdata; int *devOdata;
		int *bArray;  int *eArray; int *fArray;	int *tArray; int *dArray;
		int realN = 0;

		//Where bugs come from......
		if (n & (n - 1) != 0){
			realN = 1 << (ilog2ceil(n));
		} else {
			realN = n;
		}

		hipMalloc((void**)&devIdata, sizeof(int) * n);
		checkCUDAError("hipMalloc radix devIdata array failed");

		hipMalloc((void**)&devOdata, sizeof(int) * n);
		checkCUDAError("hipMalloc radix devOdata array failed");

		hipMalloc((void**)&bArray, sizeof(int) * n);
		checkCUDAError("hipMalloc radix bArray failed");

		hipMalloc((void**)&eArray, sizeof(int) * realN);
		hipMemset(eArray, sizeof(int) * realN, 0);
		checkCUDAError("hipMalloc radix eArray failed");

		//Remember do realN here......
		//hipMalloc((void**)&fArray, sizeof(int) * c);
		//checkCUDAError("hipMalloc radix fArray failed");

		hipMalloc((void**)&tArray, sizeof(int) * n);
		checkCUDAError("hipMalloc radix tArray failed");

		hipMalloc((void**)&dArray, sizeof(int) * n);
		checkCUDAError("hipMalloc radix dArray failed");

		int blockNum = (n + blockSize - 1) / blockSize;
		int digitNum = ilog2ceil(maxNum);

		hipMemcpy(devIdata, idata, n*sizeof(int), hipMemcpyHostToDevice);		

		//Add performance analysis
		hipEvent_t start, end;
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start);

		for (int i = 0; i < digitNum; i++) {
			kernTestTrueFalseOnRightKthBit << < blockNum, blockSize >> >(n, i, bArray, devIdata);
			kernNotOperatorOnArray << < blockNum, blockSize >> >(n, eArray, bArray);

			fArray = eArray;
			//hipMemcpy(fArray, eArray, n*sizeof(int), hipMemcpyDeviceToDevice);			
			StreamCompaction::Efficient::scanInDevice(realN, fArray);

			//Slides....
			kernComputeTArray << < blockNum, blockSize >> >(n, bArray, fArray, tArray);
			kernComputeDArray << < blockNum, blockSize >> >(n, dArray, bArray, fArray, tArray);
			kernReshuffle << <blockNum, blockSize >> >(n, devIdata, devOdata, dArray);
			std::swap(devOdata, devIdata);
		}

		//Add performance analysis
		hipEventRecord(end);
		hipEventSynchronize(end);
		float deltaTime;
		hipEventElapsedTime(&deltaTime, start, end);
		printf("GPU Radix Sort time is %f ms\n", deltaTime);

		hipMemcpy(idata, devIdata, n * sizeof(int), hipMemcpyDeviceToHost);
		hipFree(devIdata); hipFree(devOdata); hipFree(bArray); hipFree(eArray);
		/*hipFree(fArray);*/ hipFree(tArray); hipFree(dArray);

	}


}
}
