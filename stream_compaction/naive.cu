#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

// TODO: __global__
__global__ void NaiveGPUScan(int n, int *odata, const int *idata,int step)
{
	int parallelCount = threadIdx.x+blockIdx.x*blockDim.x;
	
    if(parallelCount<n)
	{
	    if(parallelCount>=step)
		{
		    odata[parallelCount]=idata[parallelCount-step]+idata[parallelCount];
		}
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
 void scan(int n, int *odata, const int *idata) {
    // TODO
  time_t start = clock();


  int* tempArray_1;
  int* tempArray_2;
  int tempCount=0;
  int step=0;

  hipMalloc((void**)&tempArray_1, n * sizeof(int));
  hipMalloc((void**)&tempArray_2, n * sizeof(int));

  //allocate the device space
  hipMemcpy(tempArray_1, idata, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(tempArray_2, idata, n * sizeof(int), hipMemcpyHostToDevice);
  


  for (tempCount = 1; tempCount <= ilog2ceil(n); ++tempCount) {
    step=2^(tempCount-1);

    NaiveGPUScan << <n, BLOCK_SIZE >> >(n, (tempCount % 2) == 0 ? tempArray_1 : tempArray_2,  (tempCount % 2) == 0 ? tempArray_2 : tempArray_1,step);
  }

  if (ilog2ceil(n) % 2 == 0) {
    Common::inclusiveToExclusive << <n, BLOCK_SIZE >> >(n, tempArray_2, tempArray_1);

    hipMemcpy(odata, tempArray_2, n * sizeof(int), hipMemcpyDeviceToHost);
  } else {
    Common::inclusiveToExclusive << <n, BLOCK_SIZE >> >(n, tempArray_1, tempArray_2);

	hipMemcpy(odata, tempArray_1, n * sizeof(int), hipMemcpyDeviceToHost);
  }

  	 time_t end = clock();
	 printf("The running time is: %f ms. \n", double(end-start)*1000/CLOCKS_PER_SEC);
    hipFree(tempArray_1);
    hipFree(tempArray_2);
}

}
}
