#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
namespace Naive {

	__global__ void kernRunScan(int N, int pow2d, int* odata, int* idata) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index > N || index < 0) {
			return;
		}
		
		if (index >= pow2d) {
			odata[index] = idata[index - pow2d] + idata[index];
		}
		else {
			odata[index] = idata[index];
		}
	}

	__global__ void kernInclusiveToExclusive(int N, int* odata, int* idata) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index > N) {
			return;
		}

		if (index == 0) {
			odata[0] = 0;
		}
		else {
			odata[index] = idata[index - 1];
		}
	}
/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	// Move data GPU-side
	int* dev_in;
	int* dev_out;

	hipMalloc((void**)&dev_in, n * sizeof(int));
	checkCUDAError("hipMalloc Error dev_in.");

	hipMalloc((void**)&dev_out, n * sizeof(int));
	checkCUDAError("hipMalloc Error dev_out.");

	hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	
	int max_d = ilog2ceil(n);

	// Loop over data 
	for (int d = 1; d <= max_d; d++) {

		kernRunScan << < fullBlocksPerGrid, threadsPerBlock >> >(n, pow(2, d - 1), dev_out, dev_in);

		int* swap = dev_out;
		dev_out = dev_in;
		dev_in = swap;
	}

	kernInclusiveToExclusive << < fullBlocksPerGrid, threadsPerBlock >> >(n, dev_out, dev_in);

	hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy back failed!");

	hipFree(dev_in);
	hipFree(dev_out);
}

}
}
