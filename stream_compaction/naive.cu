#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

__global__ void sum(int n, int startIndex, int *odata, const int *idata) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= n) return;
	if (index >= startIndex) {
		odata[index] = idata[index - startIndex] + idata[index];
	}
	else {
		odata[index] = idata[index];
	}
}

__global__ void inclusiveToExclusiveScan(int n, int *odata, const int *idata) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < n) {
		odata[index] = index == 0 ? 0 : idata[index - 1];
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
float scan(int n, int *odata, const int *idata) {
	int blockSize = 128;
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

	int* dev_idata;
	int* dev_odata;
	hipMalloc((void**)&dev_idata, n * sizeof(int));
	checkCUDAError("hipMalloc dev_idata failed!");
	
	hipMalloc((void**)&dev_odata, n * sizeof(int));
	checkCUDAError("hipMalloc dev_odata failed!");

	hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(dev_odata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	
	int numLevels = ilog2ceil(n);
	for (int startIndex = 1; startIndex <= (1 << (numLevels - 1)); startIndex *= 2) {
		sum << <fullBlocksPerGrid, blockSize >> >(n, startIndex, dev_odata, dev_idata);
		std::swap(dev_idata, dev_odata);
	}

	inclusiveToExclusiveScan << <fullBlocksPerGrid, blockSize >> >(n, dev_odata, dev_idata);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_odata);

	return milliseconds;
}

}
}
