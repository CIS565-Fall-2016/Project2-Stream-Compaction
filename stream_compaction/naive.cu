#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "common.h"
#include "naive.h"

namespace StreamCompaction {
	namespace Naive {

 
		//__global__
		__global__ void scan(int offset, int n, int *odata, const int *idata) {
			int index = threadIdx.x + blockIdx.x*blockDim.x;
			if (index >= n) return;

			if (index >= offset){
				odata[index] = idata[index] + idata[index - offset];
			}
			else{
				odata[index] = idata[index];
			}
		}
		__global__ void excludesiveShift(int n, int *odata, int *idata){
			int index = threadIdx.x + blockIdx.x* blockDim.x;
			if (index >= n) return;
			if (index >= 1){
				odata[index] = idata[index - 1];
			}
			else {
				odata[index] = 0;
			}
		}
		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		float scan(int n, int *odata, const int *idata) {
			hipEvent_t t0, t2;
			hipEventCreate(&t0);
			hipEventCreate(&t2);

			float milliscs = 0.0f;
			float tmpt;
			//dim3 numblocks(std::ceil((double) n / blockSize));
			dim3 numblocks((n + blockSize - 1) / blockSize);
			int* idata_buff;
			int* odata_buff;

			hipMalloc((void**)&idata_buff, n*sizeof(int));
			checkCUDAError("hipMalloc-idata_buff-  failed!");
			hipMalloc((void**)&odata_buff, n*sizeof(int));
			checkCUDAError("hipMalloc-odata_buff-failed!");

			/// CPU -->GPU
			hipMemcpy(idata_buff, idata, sizeof(int)*n, hipMemcpyHostToDevice);
			hipMemcpy(odata_buff, idata, sizeof(int)*n, hipMemcpyHostToDevice);


			for (int level = 1; level <= ilog2ceil(n); level++) {
				int offset;
				if (level == 1){
					offset = 1;
				}
				else {
					offset = 2 << (level - 2);
				}

				hipEventRecord(t0);
				// for the given level, all threads read from idata_buff
				scan << <numblocks, blockSize >> >(offset, n, odata_buff, idata_buff);
				hipEventRecord(t2);
				hipEventSynchronize(t2);
				hipEventElapsedTime(&tmpt, t0, t2);
				milliscs += tmpt;

				//std::swap(idata_buff, odata_buff);
				// odata_buff --> idata_buff for next iteration
				hipMemcpy(idata_buff, odata_buff, sizeof(int)*n, hipMemcpyDeviceToDevice);
			}

			hipEventRecord(t0);
			excludesiveShift << <numblocks, blockSize >> >(n, odata_buff, idata_buff);
			hipEventRecord(t2);
			hipEventSynchronize(t2);
			hipEventElapsedTime(&tmpt, t0, t2);
			milliscs += tmpt;

			//GPU --> CPU 	
			hipMemcpy(odata, odata_buff, sizeof(int)*n, hipMemcpyDeviceToHost);
			hipFree(idata_buff);
			hipFree(odata_buff);

			return milliscs;
		}

	}
}