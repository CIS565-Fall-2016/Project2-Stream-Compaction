#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
namespace Naive {

	__global__ void kernRunScan(int N, int pow2d, int* odata, int* idata) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;

		if (index > N) {
			return;
		}
		
		if (index >= pow2d) {
			odata[index] = idata[index - pow2d] + idata[index];
		}
		else {
			odata[index] = idata[index];
		}
	}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
	dim3 threadsPerBlock(blockSize);

	// Move data GPU-side
	int* dev_in;
	int* dev_out;

	hipMalloc((void**)&dev_in, n * sizeof(int));
	checkCUDAError("hipMalloc Error dev_a.");

	hipMalloc((void**)&dev_out, n * sizeof(int));
	checkCUDAError("hipMalloc Error dev_b.");

	hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	
	int max_d = ilog2ceil(n);

	// Loop over data 
	for (int d = 1; d < max_d; d++) {

		kernRunScan <<< fullBlocksPerGrid, threadsPerBlock >>>(n, pow(2, d - 1), dev_out, dev_in);

	}	

	hipMemcpy(odata, dev_out, sizeof(int) * n, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy back failed!");

	hipFree(dev_in);
	hipFree(dev_out);
}

}
}
