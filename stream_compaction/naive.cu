#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define BLOCK_SIZE 128
#define BLOCK_COUNT(n) ((n + BLOCK_SIZE - 1) / BLOCK_SIZE)

namespace StreamCompaction {
namespace Naive {

// TODO: __global__

__global__ void naiveScan(int n, int offset, int* odata, const int *idata) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  if (tid >= n) {
    return;
  }
  if (tid >= offset) {
    odata[tid] = idata[tid - offset] + idata[tid] ;
  } else {
    odata[tid] = idata[tid];
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO

  int* dev_odata1;
  int* dev_odata2;

  hipMalloc((void**)&dev_odata1, n * sizeof(int));
  hipMalloc((void**)&dev_odata2, n * sizeof(int));

  hipMemcpy(dev_odata1, idata, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_odata2, idata, n * sizeof(int), hipMemcpyHostToDevice);

  int height = ilog2ceil(n);
  //height = 2;
  for (int level = 1; level <= height; ++level) {
    int offset = pow(2, level - 1);
    naiveScan << <BLOCK_COUNT(n), BLOCK_SIZE >> >(
      n, 
      offset, 
      (level % 2) == 0 ? dev_odata1 : dev_odata2, 
      (level % 2) == 0 ? dev_odata2 : dev_odata1
        );
  }

  if (height % 2 == 0) {
    Common::inclusiveToExclusiveScanResult << <BLOCK_COUNT(n), BLOCK_SIZE >> >(n, dev_odata2, dev_odata1);
    hipMemcpy(odata, dev_odata2, n * sizeof(int), hipMemcpyDeviceToHost);
  } else {
    Common::inclusiveToExclusiveScanResult << <BLOCK_COUNT(n), BLOCK_SIZE >> >(n, dev_odata1, dev_odata2);
    hipMemcpy(odata, dev_odata1, n * sizeof(int), hipMemcpyDeviceToHost);
  }

  hipFree(dev_odata1);
  hipFree(dev_odata2);
}

}
}
