#include "hip/hip_runtime.h"
#include "naive.h"

#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

namespace StreamCompaction {
namespace Naive {

// DONE: __global__
__global__ void kernNaiveScanPass(int N, int offset, int* in_buffer, int* out_buffer)
{
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= N) { return; }

    if (index >= offset)
    {
        out_buffer[index] = in_buffer[index - offset] + in_buffer[index];
    }
    else
    {
        out_buffer[index] = in_buffer[index];
    }
}

int getNaiveScanMaxPotentialBlockSize()
{
    // not thread-safe
    static int block_size = -1;
    if (block_size == -1)
    {
        block_size = calculateBlockSizeForDeviceFunction(kernNaiveScanPass);
    }
    return block_size;
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) 
{
    if (n <= 0) { return; }
    if (n == 1) { odata[0] = idata[0]; return; }

    auto block_size = getNaiveScanMaxPotentialBlockSize();
    auto full_blocks_per_grid = (n + block_size - 1) / block_size;

    // DONE
    int* dev_in_buffer;
    hipMalloc((void**)&dev_in_buffer, n * sizeof(*dev_in_buffer));
    checkCUDAError("hipMalloc dev_in_buffer failed!");
    int* dev_out_buffer;
    hipMalloc((void**)&dev_out_buffer, n * sizeof(*dev_out_buffer));
    checkCUDAError("hipMalloc dev_out_buffer failed!");

    hipMemcpy(dev_in_buffer, idata, n * sizeof(*idata), hipMemcpyHostToDevice);
    
    auto cap = ilog2ceil(n);
    int offset;
    for (int d = 1; d <= cap; d++)
    {
        offset = 1 << (d - 1);
        kernNaiveScanPass <<< full_blocks_per_grid, block_size >>>(n, offset, dev_in_buffer, dev_out_buffer);
        std::swap(dev_in_buffer, dev_out_buffer);
    }
    std::swap(dev_in_buffer, dev_out_buffer);

    hipMemcpy(odata, dev_out_buffer, n * sizeof(*odata), hipMemcpyDeviceToHost);

    hipFree(dev_in_buffer);
    hipFree(dev_out_buffer);
}

}
}
