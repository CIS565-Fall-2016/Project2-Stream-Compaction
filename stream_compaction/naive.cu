#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

	__global__ void parallelAdd(int n, int depth, int* odata, int* idata)
{
	int k = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (k >= n)
		return;

	if (k < (int)powf(2, depth - 1))
		odata[k] = idata[k];
	else
		odata[k] = idata[k - (int)powf(2, depth - 1)] + idata[k];
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    
	dim3 numBlocks = (n + blocksize - 1) / blocksize;

	int * dev_beforeScan;
	int * dev_afterScan;

	hipMalloc((void **)&dev_beforeScan, n * sizeof(int));
	checkCUDAError("hipMalloc dev_beforeScan failed!");
	hipMalloc((void **)&dev_afterScan, n * sizeof(int));
	checkCUDAError("hipMalloc dev_afterScan failed!");

	hipMemcpy(dev_beforeScan, idata, sizeof(int) * n, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy idata to dev_afterScan failed!");

	//for d = 1 to lg(n) do
	for (int depth = 1; depth <= ilog2ceil(n); ++depth)
	{
		parallelAdd << <numBlocks, blocksize >> >(n, depth, dev_afterScan, dev_beforeScan);

		//ping-pong buffers
		int * temp = dev_afterScan;
		dev_afterScan = dev_beforeScan;
		dev_beforeScan = temp;
	}

	//because of ping-ponging, last iteration will be stored in beforeScan
	//additionally, we need to convert the inclusive scan into an exclusive scan
	odata[0] = 0;
	hipMemcpy(&odata[1], dev_beforeScan, sizeof(int) * (n-1), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy dev_beforeScan to odata failed!");

	hipFree(dev_beforeScan);
	hipFree(dev_afterScan);
}

}
}


