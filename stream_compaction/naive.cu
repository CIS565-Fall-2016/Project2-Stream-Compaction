#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "timer.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

	__global__ void kernSumUp(int N, int inStartIdx, int* inArray, int* outArray)
	{
		const int iSelf = threadIdx.x + (blockIdx.x * blockDim.x);
		if (inStartIdx <= iSelf && iSelf < N)
		{
			outArray[iSelf] = inArray[iSelf - inStartIdx] + inArray[iSelf];
		}
	}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
	void scan(int n, int *odata, const int *idata)
	{
		if (n <= 0 || odata == NULL || idata == NULL)
			return;

		const int blockSize = 96;
		dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

		int* cudaIn = NULL;
		hipMalloc((void**)&cudaIn, n * sizeof(int));
		checkCUDAErrorWithLine("hipMalloc cudaIn failed!");
		hipMemcpy(cudaIn, idata, n * sizeof(int), hipMemcpyHostToDevice);

		int* cudaOut = NULL;
		hipMalloc((void**)&cudaOut, n * sizeof(int));
		checkCUDAErrorWithLine("hipMalloc cudaOut failed!");

		int maxDepth = ilog2ceil(n);
		Timer::playTimer();
		for (int d = 0; d < maxDepth; ++d)
		{
			const int inStartIdx = 1 << d;
			hipMemcpy(cudaOut, cudaIn, inStartIdx * sizeof(int), hipMemcpyHostToDevice);

			kernSumUp << <fullBlocksPerGrid, blockSize >> >(n, inStartIdx, cudaIn, cudaOut);

			// Ping-pong the buffers
			int* cudaTemp = cudaIn; cudaIn = cudaOut; cudaOut = cudaTemp;
		}
		Common::convertInclusiveToExclusiveScan << <fullBlocksPerGrid, blockSize >> > (n, cudaIn, cudaOut);
		Timer::pauseTimer();

		hipMemcpy(odata, cudaOut, n * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(cudaOut);
		hipFree(cudaIn);
	}

}
}
