#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

// TODO: __global__

__global__ void naiveScan(int n, int offset, int* odata, const int *idata) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  if (tid >= n) {
    return;
  }
  if (tid >= offset) {
    odata[tid] = idata[tid - offset] + idata[tid] ;
  } else {
    odata[tid] = idata[tid];
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int* dev_odata1;
  int* dev_odata2;

  hipMalloc((void**)&dev_odata1, n * sizeof(int));
  hipMalloc((void**)&dev_odata2, n * sizeof(int));

  hipMemcpy(dev_odata1, idata, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_odata2, idata, n * sizeof(int), hipMemcpyHostToDevice);
  
  hipEventRecord(start);
  int height = ilog2ceil(n);
  for (int level = 1; level <= height; ++level) {
    int offset = 1 << (level - 1);
    naiveScan << <BLOCK_COUNT(n), BLOCK_SIZE >> >(
      n, 
      offset, 
      (level % 2) == 0 ? dev_odata1 : dev_odata2, 
      (level % 2) == 0 ? dev_odata2 : dev_odata1
        );
  }

  if (height % 2 == 0) {
    Common::inclusiveToExclusiveScanResult << <BLOCK_COUNT(n), BLOCK_SIZE >> >(n, dev_odata2, dev_odata1);
	hipEventRecord(stop);
    hipMemcpy(odata, dev_odata2, n * sizeof(int), hipMemcpyDeviceToHost);
  } else {
    Common::inclusiveToExclusiveScanResult << <BLOCK_COUNT(n), BLOCK_SIZE >> >(n, dev_odata1, dev_odata2);
	hipEventRecord(stop);
	hipMemcpy(odata, dev_odata1, n * sizeof(int), hipMemcpyDeviceToHost);
  }

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Runtime: %d ns\n", (int)MS_TO_NS(milliseconds));

  hipFree(dev_odata1);
  hipFree(dev_odata2);
}

}
}
