#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <algorithm>

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
	namespace Naive {

		// TODO: __global__

		__global__ void kernReduce(int offset, int n, int *in, int *out) {
			int index = threadIdx.x + (blockDim.x * blockIdx.x);

			if (index >= n) return;

			if (index >= offset) {
				out[index] = in[index] + in[index - offset];
			}
			else {
				out[index] = in[index];
			}

		}

		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int *dev_out;
			int *dev_in;

			hipMalloc((void**)&dev_in, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_in failed!");
			hipMalloc((void**)&dev_out, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_out failed!");

			hipMemcpy(dev_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			for (int d = 1; d <= ilog2ceil(n); ++d) {
				kernReduce << <fullBlocksPerGrid, blockSize >> >((1 << (d - 1)), n, dev_in, dev_out);
				std::swap(dev_in, dev_out);
			}
			std::swap(dev_in, dev_out);
			hipMemcpy(odata + 1, dev_out, sizeof(int) * (n-1), hipMemcpyDeviceToHost);
			
			/*for (int i = n-1; i > 0; --i) {
				odata[i] = odata[i - 1];
			}*/
			
			odata[0] = 0;
			//printf("TODO\n");
			hipFree(dev_out);
			hipFree(dev_in);
		}

	}
}
