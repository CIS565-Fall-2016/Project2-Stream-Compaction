#include "hip/hip_runtime.h"
#include "naive.h"

#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

namespace StreamCompaction {
namespace Naive {

    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

// DONE: __global__
__global__ void kernNaiveScanPass(int N, int offset, int* in_buffer, int* out_buffer)
{
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= N) { return; }

    if (index >= offset)
    {
        out_buffer[index] = in_buffer[index - offset] + in_buffer[index];
    }
    else
    {
        out_buffer[index] = in_buffer[index];
    }
}

int getNaiveScanBlockSize()
{
    // not thread-safe
    static int block_size = -1;
    if (block_size == -1)
    {
        block_size = calculateBlockSizeForDeviceFunction(kernNaiveScanPass);
    }
    return block_size;
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) 
{
    if (n <= 0) { return; }

    auto block_size = getNaiveScanBlockSize();
    auto full_blocks_per_grid = (n + block_size - 1) / block_size;

    // DONE
    int* dev_in_buffer;
    hipMalloc((void**)&dev_in_buffer, n * sizeof(*dev_in_buffer));
    checkCUDAError("hipMalloc dev_in_buffer failed!");
    int* dev_out_buffer;
    hipMalloc((void**)&dev_out_buffer, n * sizeof(*dev_out_buffer));
    checkCUDAError("hipMalloc dev_out_buffer failed!");

    hipMemcpy(dev_in_buffer, idata, n * sizeof(*idata), hipMemcpyHostToDevice);
    
    timer().startGpuTimer();

    auto cap = ilog2ceil(n);
    int offset;
    for (int d = 1; d <= cap; d++)
    {
        offset = 1 << (d - 1);
        kernNaiveScanPass <<< full_blocks_per_grid, block_size >>>(n, offset, dev_in_buffer, dev_out_buffer);
        std::swap(dev_in_buffer, dev_out_buffer);
    }
    std::swap(dev_in_buffer, dev_out_buffer);

    timer().endGpuTimer();

    // defered copy because of exclusive scan
    hipMemcpy(odata + 1, dev_out_buffer, (n - 1) * sizeof(*odata), hipMemcpyDeviceToHost);
    odata[0] = 0;

    hipFree(dev_in_buffer);
    hipFree(dev_out_buffer);
}

}
}
