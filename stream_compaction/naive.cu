#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
namespace Naive {

// TODO: __global__

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
int threadPerBlock = 1024;
int BlockNum;

int *dev_Data[2];

__global__ void CudaScan(int d, int *in, int *out, int n)
{
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thid >= n)
		return;
	int m = 1 << (d - 1);
		
	if (thid >= m)
		out[thid] = in[thid] + in[thid - m];
	else
		out[thid] = in[thid];

}

void scan(int n, int *odata, const int *idata) {

	int nCeilLog = ilog2ceil(n);
	int nLength = 1 << nCeilLog;

	hipMalloc((void**)&dev_Data[0], nLength * sizeof(int));
	hipMalloc((void**)&dev_Data[1], nLength * sizeof(int));
	checkCUDAError("hipMalloc failed!");

	hipMemcpy(dev_Data[0], idata, sizeof(int) * nLength, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy to device failed!");

	int nOutputIndex = 0;
	float time_elapsed=0;
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord( start,0);
	for (int i = 1; i <= nCeilLog; i++)
	{
		nOutputIndex ^= 1;
		BlockNum = nLength / threadPerBlock + 1;
		CudaScan<<<BlockNum, threadPerBlock>>>(i, dev_Data[nOutputIndex ^ 1], dev_Data[nOutputIndex], nLength);
	}
		hipEventRecord( stop,0);
hipEventSynchronize(start);
hipEventSynchronize(stop);
hipEventElapsedTime(&time_elapsed,start,stop);
//FILE* fp = fopen("efficient.txt", "a+");
//fprintf(fp, "%d %f\n", nCeilLog, time_elapsed);
//fclose(fp);
	odata[0] = 0;
	hipMemcpy(odata + 1, dev_Data[nOutputIndex], sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy to host failed!");	

	hipFree(dev_Data[0]);
	hipFree(dev_Data[1]);
}



}
}
