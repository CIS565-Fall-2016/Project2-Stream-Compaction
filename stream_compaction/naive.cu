#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
	namespace Naive {

		// TODO: __global__ : finished

		__global__ void kernScan(int N, int start_idx, int *odata, const int *idata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= N) return;
			if (index >= start_idx)
			{
				odata[index] = idata[index - start_idx] + idata[index];
			}
			else
			{
				odata[index] = idata[index];
			}
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		float scan(int n, int *odata, const int *idata, int blockSize) {
			// TODO : finished
			// record time
			float diff(0);
			hipEvent_t start, end;
			hipEventCreate(&start);
			hipEventCreate(&end);
			hipEventRecord(start, 0);

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			int *tmp_data, *tmp_data2;
			hipMalloc((void**)&tmp_data, n * sizeof(int));
			hipMalloc((void**)&tmp_data2, n * sizeof(int));
			hipMemset(tmp_data2, 0, n * sizeof(int));
			hipMemset(tmp_data, 0, n * sizeof(int));
			hipMemcpy(tmp_data+1, idata, (n-1) * sizeof(int), hipMemcpyHostToDevice);
			int loop_times = ilog2ceil(n);
			int start_idx = 1;
			for (int i = 0; i < loop_times; ++i)
			{
				kernScan<<<fullBlocksPerGrid, blockSize>>>(n, start_idx, tmp_data2, tmp_data);
				int *tmp_pt = tmp_data;
				tmp_data = tmp_data2;
				tmp_data2 = tmp_pt;
				start_idx *= 2;
			}

			hipMemcpy(odata, tmp_data, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(tmp_data);
			hipFree(tmp_data2);

			hipEventRecord(end, 0);
			hipEventSynchronize(start);
			hipEventSynchronize(end);
			hipEventElapsedTime(&diff, start, end);

			//printf("GPU naive scan took %fms\n", diff);
			return diff;
		}

	}
}
