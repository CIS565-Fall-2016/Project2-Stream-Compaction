#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

int* devIdata;
int* devOdata;

namespace StreamCompaction {
	namespace Naive {

		// TODO: __global__
		__global__ void kernelNaive(int n, int delta, const int *idata, int *odata) {
			int index = (blockIdx.x *blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			if (index - delta < 0) {
				odata[index] = idata[index];
			} else {
               	odata[index] = idata[index - delta] + idata[index];
			}
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {
			// TODO
			// printf("TODO\n");
			hipMalloc((void**)&devIdata, n * sizeof(int));
			checkCUDAError("hipMalloc devIdata failed");

			hipMalloc((void**)&devOdata, n * sizeof(int));
			checkCUDAError("hipMalloc devOdata failed");

			hipMemcpy(devIdata, idata, n*sizeof(int), hipMemcpyHostToDevice);
			
			//performance check... remember...

			int blockNum = (n + blockSize - 1) / blockSize;
			
			//Naive Parallel Scan
			int level = ilog2ceil(n);
			int delta;
			for (int d = 1; d <= level; d++) {
				// pow (2,d-1)
				// refer to slides 
				delta = (1 << (d - 1));
				kernelNaive << < blockNum, blockSize >> >(n, delta, devIdata, devOdata);
				std::swap(devIdata, devOdata);
			}
			// Think twice.............
			std::swap(devIdata, devOdata);
			
			// exclusive scan, set odata[0] = 0 seperately 
			hipMemcpy(odata + 1, devOdata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
			odata[0] = 0;

			hipFree(devIdata);
			hipFree(devOdata);

			checkCUDAError("naice scan error...");
		}

	}
}
