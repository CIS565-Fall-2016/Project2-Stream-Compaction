#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

int* devIdata;
int* devOdata;

namespace StreamCompaction {
	namespace Naive {

		// TODO: __global__
		__global__ void kernelNaive(int n, int delta, const int *idata, int *odata) {
			int index = (blockIdx.x *blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			if (index - delta < 0) {
				odata[index] = idata[index];
			} else {
               	odata[index] = idata[index - delta] + idata[index];
			}
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int *odata, const int *idata) {
			// TODO
			// printf("TODO\n");
			hipMalloc((void**)&devIdata, n * sizeof(int));
			checkCUDAError("hipMalloc devIdata failed");

			hipMalloc((void**)&devOdata, n * sizeof(int));
			checkCUDAError("hipMalloc devOdata failed");

			hipMemcpy(devIdata, idata, n*sizeof(int), hipMemcpyHostToDevice);
			
			//performance check... remember...

			int blockNum = (n + blockSize - 1) / blockSize;
			

			//Add performance analysis
			hipEvent_t start, end;
			hipEventCreate(&start);
			hipEventCreate(&end);
			hipEventRecord(start);

			//Naive Parallel Scan
			int level = ilog2ceil(n);
			int delta;
			for (int d = 1; d <= level; d++) {
				// pow (2,d-1)
				// refer to slides 
				delta = (1 << (d - 1));
				kernelNaive << < blockNum, blockSize >> >(n, delta, devIdata, devOdata);
				std::swap(devIdata, devOdata);
			}
			// Think twice.............
			std::swap(devIdata, devOdata);
			//Add performance analysis
			hipEventRecord(end);
			hipEventSynchronize(end);
			float deltaTime;
			hipEventElapsedTime(&deltaTime, start, end);
			printf("GPU Naive Scan time is %f ms\n", deltaTime);
			
			// exclusive scan, set odata[0] = 0 seperately 
			hipMemcpy(odata + 1, devOdata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);
			odata[0] = 0;

			hipFree(devIdata);
			hipFree(devOdata);

			checkCUDAError("naice scan error...");
		}

	}
}
