#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
namespace Naive {

__global__ void kernNaiveScan(int n, int round, int * odata, int * idata) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) {
		return;
	}
	odata[index] = (
		(index < (1 << (round - 1))) 
			? 0 
			: idata[index - (1 << (round - 1))]
		) + idata[index];
}

__global__ void kernInclusiveToExclusiveScan(int n, int * odata, int * idata) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= n) {
		return;
	}
	odata[index] = (index == 0 ) ? 0 : idata[index - 1];
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

	int * dev_data;
	int * dev_data2;
	hipMalloc((void**)&dev_data, n * sizeof(int));
	hipMalloc((void**)&dev_data2, n * sizeof(int));
	hipMemcpy((void*)dev_data, (void*)idata, n * sizeof(int), hipMemcpyHostToDevice);

#if TIMING == 1
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif

	for (int i = 1; i <= ilog2ceil(n); i++) {
		kernNaiveScan << <fullBlocksPerGrid, blockSize >> >(n, i, dev_data2, dev_data);
		int * tempPtr = dev_data;
		dev_data = dev_data2;
		dev_data2 = tempPtr;
	}
	kernInclusiveToExclusiveScan << <fullBlocksPerGrid, blockSize >> >(n, dev_data2, dev_data);

#if TIMING == 1
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Naive scan: %f milliseconds\n", milliseconds);
#endif

	hipMemcpy((void*)odata, (void*)dev_data2, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_data);
	hipFree(dev_data2);
}

}
}
