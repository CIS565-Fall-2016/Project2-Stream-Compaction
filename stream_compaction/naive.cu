#include "hip/hip_runtime.h"
#include "naive.h"

#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

namespace StreamCompaction {
namespace Naive {


// TODO: __global__
__global__ void kernScanPass(int N, int offset, int* in_buffer, int* out_buffer)
{
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= N) { return; }

    if (index >= offset)
    {
        out_buffer[index] = in_buffer[index - offset] + in_buffer[index];
    }
    else
    {
        out_buffer[index] = in_buffer[index];
    }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) 
{
    int block_size;
    int min_grid_size; 
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, kernScanPass);
    auto full_blocks_per_grid = (n + block_size - 1) / block_size;

    // TODO
    int* dev_in_buffer;
    hipMalloc((void**)&dev_in_buffer, n * sizeof(*dev_in_buffer));
    checkCUDAError("hipMalloc dev_in_buffer failed!");
    int* dev_out_buffer;
    hipMalloc((void**)&dev_out_buffer, n * sizeof(*dev_out_buffer));
    checkCUDAError("hipMalloc dev_out_buffer failed!");

    hipMemcpy(dev_in_buffer, idata, n * sizeof(*idata), hipMemcpyHostToDevice);
    
    auto cap = ilog2ceil(n);
    int offset;
    for (int d = 1; d <= cap; d++)
    {
        offset = 1 << (d - 1);
        kernScanPass <<< full_blocks_per_grid, block_size >>>(n, offset, dev_in_buffer, dev_out_buffer);
        std::swap(dev_in_buffer, dev_out_buffer);
    }

    std::swap(dev_in_buffer, dev_out_buffer);
    hipMemcpy(odata, dev_out_buffer, n * sizeof(*odata), hipMemcpyDeviceToHost);

    hipFree(dev_in_buffer);
    hipFree(dev_out_buffer);
}

}
}
