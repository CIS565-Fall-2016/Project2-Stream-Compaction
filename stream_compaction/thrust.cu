#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    thrust::device_vector<int> dv_idata(idata, idata + n);
	thrust::device_vector<int> dv_odata(odata, odata + n);

	thrust::exclusive_scan(dv_idata.begin(), dv_idata.end(), dv_odata.begin());	
	
	thrust::copy(dv_odata.begin(), dv_odata.end(), odata);
}

}
}
