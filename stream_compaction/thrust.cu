#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
	thrust::host_vector<int> thrust_odata(odata, odata + n);
	thrust::host_vector<int> thrust_idata(idata, idata + n);

	thrust::device_vector<int> dev_thrust_odata = thrust_odata;
	thrust::device_vector<int> dev_thrust_idata = thrust_idata;

	thrust::exclusive_scan(dev_thrust_idata.begin(), dev_thrust_idata.end(), dev_thrust_odata.begin());

	thrust::copy(dev_thrust_odata.begin(), dev_thrust_odata.end(), odata);
}

}
}
