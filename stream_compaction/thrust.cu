#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	thrust::device_vector<int> dv_in(idata, idata + n);
	thrust::device_vector<int> dv_out(odata, odata + n);

    thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
	thrust::copy(dv_out.begin(), dv_out.end(), odata);
}

}
}
