#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	thrust::device_vector<int> dev_thrust_idata(idata, idata + n);
	thrust::device_vector<int> dev_thrust_odata(n);

#if TIMING == 1
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif

	thrust::exclusive_scan(dev_thrust_idata.begin(), dev_thrust_idata.end(), dev_thrust_odata.begin());

#if TIMING == 1
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Thrust scan: %f milliseconds\n", milliseconds);
#endif

	thrust::copy(dev_thrust_odata.begin(), dev_thrust_odata.end(), odata);
}

}
}
