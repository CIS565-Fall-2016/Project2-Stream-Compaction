#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/*int *dev_idata;*/
int *dev_odata;

thrust::device_vector<int> dev_thrust_idata;
thrust::device_vector<int> dev_thrust_odata;

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	/*
	hipMalloc((void**)&dev_idata, n * sizeof(int));
	checkCUDAError("hipMalloc dev_idata failed!");*/
	hipMalloc((void**)&dev_odata, n * sizeof(int));
	checkCUDAError("hipMalloc dev_odata failed!");
	/*hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy idata to dev_idata failed!");*/

	thrust::device_vector<int> dev_thrust_idata(idata, idata + n);
	thrust::device_vector<int> dev_thrust_odata(odata, odata + n);

	thrust::exclusive_scan(dev_thrust_idata.begin(), dev_thrust_idata.end(),
		dev_thrust_odata.begin());

	thrust::copy(dev_thrust_odata.begin(), dev_thrust_odata.end(), dev_odata);

	hipMemcpy(odata, dev_odata, n*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy dev_odata to odata failed!");
}

}
}
