#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {


	static StreamCompaction::Common::Timer timer;

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

	thrust::host_vector<int> host_input(idata, idata + n);
	thrust::device_vector<int> dev_input = host_input;

	//thrust::host_vector<int> host_output(odata, odata + n);
	thrust::device_vector<int> dev_output(odata, odata + n);

	// what happened during thrust? GPU timer malfunctioning
	timer.startGpuTimer();
	// call
	thrust::exclusive_scan(dev_input.begin(), dev_input.end(), dev_output.begin());

	timer.stopGpuTimer();

	timer.printTimerInfo("Scan::Thrust = ", timer.getGpuElapsedTime());

	thrust::copy(dev_output.begin(), dev_output.end(), odata);
}

}
}
