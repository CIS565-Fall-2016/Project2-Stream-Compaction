#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"
#include <chrono>

namespace StreamCompaction {
namespace Thrust {

int *dev_Data;
int *dev_OutputData;
/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
auto start = std::chrono::system_clock::now();
	thrust::exclusive_scan(idata, idata + n, odata);
	auto end   = std::chrono::system_clock::now();
	//auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end-start).count();
	std::cout << duration << std::endl;
	FILE* fp = fopen("efficient.txt", "a+");
	fprintf(fp, "%d %I64d\n", ilog2ceil(n), duration);
	fclose(fp);
}

}
}
