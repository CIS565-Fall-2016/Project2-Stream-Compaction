#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"
#include <ctime>

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
float scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
 

	for (int i=0;i <10000 ; i++){
	thrust::exclusive_scan(idata , idata +n , odata);
	}
 
	return 0;
}

}
}
