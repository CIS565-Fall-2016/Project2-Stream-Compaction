#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"
#include "profilingcommon.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

  // Convert to device vector
  thrust::device_vector<int> dev_idata(idata, idata + n);
  thrust::device_vector<int> dev_odata(odata, odata + n);

#ifdef PROFILE
  auto begin = std::chrono::high_resolution_clock::now();
  for (auto it = 0; it < PROFILE_ITERATIONS; ++it) {
#endif

  thrust::exclusive_scan(dev_idata.begin(), dev_idata.end(), dev_odata.begin());

#ifdef PROFILE
  }
  auto end = std::chrono::high_resolution_clock::now();
  std::cout << "Runtime: " << std::chrono::duration_cast<std::chrono::nanoseconds>((end - begin) / PROFILE_ITERATIONS).count() << " ns" << std::endl;
#endif

  thrust::host_vector<int> host_odata = dev_odata;
  hipMemcpy(odata, host_odata.data(), n * sizeof(int), hipMemcpyHostToHost);
}

}
}
