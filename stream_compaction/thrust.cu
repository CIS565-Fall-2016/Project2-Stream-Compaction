#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

    //thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());


    thrust::host_vector<int> dv_in(idata, idata + n);
    thrust::host_vector<int> dv_out(odata, odata + n);

    thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

    //odata = &(dv_out.front()); 
    for (int i = 0; i < n; i++)
        odata[i] = dv_out[i];
}

}
}
