#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "common.h"
#include "thrust.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	thrust::exclusive_scan(idata, idata + n, odata);
}

void sort(int n, int *idata) {
	thrust::sort(idata, idata + n);
}

}
}
