#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "radixSort.h"
#include "efficient.h"

namespace StreamCompaction {
	namespace RadixSort {
#define blockSize 128

		//int getNbit(int input, int nth){
		//	return (input >> nth) & 1;
		//}

		// assume the input and output are bits
		__global__ void computeE(int n, int * edata, const int * bdata){
			int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index < n) {
				//edata[index] = ~(bdata[index]);
				edata[index] = 1 - (bdata[index]);
				//if (index ==0){
				//	odata[index] = ~(0|idata[index]);
				//}
				//else {
				//	odata[index] = ~(idata[index-1]|idata[index]);
				//}
			}
		}
		__global__ void computeT(int n, int * tdata, const int * fdata, const int totalFalses){
			int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index < n){
				tdata[index] = index - fdata[index] + totalFalses;
			}
		}

		__global__ void computeB(int n, int *bdata, const int *idata, int ith){
			int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index < n){
				//bdata[index] =  (idata[index]>> ith) & 1; 
				bdata[index] = (idata[index] & (1<<ith ))>>ith;  
			}
		}

		__global__ void computeD(int n, int *ddata, const int * bdata, const int *tdata, const int * fdata){
			int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index < n){
				//printf(" %d \n",bdata[index]);
				ddata[index] = bdata[index] ? tdata[index] : fdata[index];
			}
		}

		__global__ void scatter(int n, int *odata, const int *idata, const int * ddata){
			int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index < n){
				//odata[index]= idata[ddata[index]];
				odata[ddata[index]]= idata[index];
			}
		}
		/**
		* radix sort 
		*/
		void sort(int n, int *odata, const int *idata, int msb) {
			dim3 numblocks(std::ceil((double) n / blockSize));

			int * idata_buff;
			int * idata_buff2;
			int * bdata_buff;
			int * edata_buff;
			int * fdata_buff;
			int * tdata_buff;
			int * ddata_buff;


			hipMalloc((void**)&idata_buff,n*sizeof(int));
			checkCUDAError("hipMalloc-idata_buff-  failed!");	
			hipMalloc((void**)&idata_buff2,n*sizeof(int));
			checkCUDAError("hipMalloc-idata_buff2-  failed!");	
			hipMalloc((void**)&bdata_buff,n*sizeof(int));
			checkCUDAError("hipMalloc-bdata_buff-  failed!");	
			hipMalloc((void**)&edata_buff,n*sizeof(int));
			checkCUDAError("hipMalloc-edata_buff-  failed!");	
			hipMalloc((void**)&fdata_buff,n*sizeof(int));
			checkCUDAError("hipMalloc-fdata_buff-  failed!");	
			hipMalloc((void**)&tdata_buff,n*sizeof(int));
			checkCUDAError("hipMalloc-tdata_buff-  failed!");	
			hipMalloc((void**)&ddata_buff,n*sizeof(int));
			checkCUDAError("hipMalloc-ddata_buff-  failed!");	

			/// CPU -->GPU
			hipMemcpy(idata_buff,idata,n*sizeof(int),hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy-idata_buff-failed");

			for (int i=0; i<= msb; i++){
				int totalFalses;
				int totalFalses1 = 0;
				int totalFalses2 = 0;
				//find b array for each bit
				computeB<<<numblocks, blockSize>>>(n, bdata_buff, idata_buff, i);
				computeE<<<numblocks, blockSize>>>(n,  edata_buff, bdata_buff);
				StreamCompaction::Efficient::scan(n, fdata_buff, edata_buff);

				hipMemcpy(&totalFalses1,edata_buff+n-1,sizeof(int),hipMemcpyDeviceToHost);
				hipMemcpy(&totalFalses2,fdata_buff+n-1,sizeof(int),hipMemcpyDeviceToHost);
				totalFalses = totalFalses1 + totalFalses2;

				computeT<<<numblocks, blockSize>>>(n,  tdata_buff,  fdata_buff,  totalFalses);
				computeD<<<numblocks, blockSize>>>(n, ddata_buff, bdata_buff, tdata_buff, fdata_buff);


				//scatter darray for this bit
				scatter<<<numblocks, blockSize>>>(n, idata_buff2, idata_buff, ddata_buff);
				hipMemcpy(idata_buff,idata_buff2,n*sizeof(int),hipMemcpyDeviceToDevice);
				checkCUDAError("hipMemcpy-idata_buff-failed");
			}

			//GPU --> CPU
			hipMemcpy(odata,idata_buff,n*sizeof(int),hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy-odata-failed");
			//free
			hipFree(idata_buff);
			hipFree(idata_buff2);
			hipFree(bdata_buff);
			hipFree(tdata_buff);
			hipFree(fdata_buff);
			hipFree(edata_buff);
			hipFree(ddata_buff);
		} 

	}
}
