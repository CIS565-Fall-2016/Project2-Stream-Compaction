#include "hip/hip_runtime.h"
#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}


namespace StreamCompaction {
namespace Common {

/**
 * Maps an array to an array of 0s and 1s for stream compaction. Elements
 * which map to 0 will be removed, and elements which map to 1 will be kept.
 */
__global__ void kernMapToBoolean(int n, int *bools, const int *idata) {
    // TODO
	int parallelCount = threadIdx.x+blockIdx.x*blockDim.x;
	if(parallelCount<n)
	{
	    if(idata[parallelCount]==0)
	{
	    bools[parallelCount]=0;
	}
	else{
	    bools[parallelCount]=1;
	}
	}
	
}

/**
 * Performs scatter on an array. That is, for each element in idata,
 * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
 */
__global__ void kernScatter(int n, int *odata,
        const int *idata, const int *bools, const int *indices) {
    // TODO
	int parallelCount = threadIdx.x+blockIdx.x*blockDim.x;
	if(bools[parallelCount]==1)
	{
	    odata[indices[parallelCount]]=idata[parallelCount];
	}
}

__global__ void inclusiveToExclusive(int n, int *idata, int *odata)
{
    int parallelCount = threadIdx.x+blockIdx.x*blockDim.x;
	if(parallelCount<n)
	{
	    if(parallelCount == 0)
		{
		    odata[0]=0;
			return;
		}

		odata[parallelCount] = idata[parallelCount-1];
	}
}

}
}
